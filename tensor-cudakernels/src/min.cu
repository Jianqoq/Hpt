
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <limits.h>
#define WRAP 32

#define BOOL_MAX 1
#define I8_MAX SCHAR_MAX
#define I16_MAX SHRT_MAX
#define I32_MAX INT_MAX
#define I64_MAX LLONG_MAX
#define F32_MAX INFINITY
#define F64_MAX INFINITY

#define U8_MAX UCHAR_MAX
#define U16_MAX USHRT_MAX
#define U32_MAX UINT_MAX
#define U64_MAX ULONG_MAX
#define F16_MAX __half((unsigned short)31744)
#define BF16_MAX __hip_bfloat16((unsigned short)0x7F80)

#define min_bool(a, b) ((bool)min(((unsigned char)a), ((unsigned char)b)))
#define min_i8(a, b) min((a), (b))
#define min_i16(a, b) min((a), (b))
#define min_i32(a, b) min((a), (b))
#define min_i64(a, b) min((a), (b))
#define min_f32(a, b) min((a), (b))
#define min_f64(a, b) min((a), (b))

#define min_u8(a, b) min((a), (b))
#define min_u16(a, b) min((a), (b))
#define min_u32(a, b) min((a), (b))
#define min_u64(a, b) min((a), (b))

#define min_f16(a, b) __hmin((a), (b))
#define min_bf16(a, b) __hmin((a), (b))

#define DEFINE_REDUCE_KERNEL(rust_type, type, INIT_VAL)                                                                                                                       \
    __device__ __forceinline__ void warpReduce_##rust_type(volatile type *sdata_##rust_type, unsigned int tid)                                                                \
    {                                                                                                                                                                         \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]);                                                                        \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]);                                                                        \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]);                                                                         \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]);                                                                         \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]);                                                                         \
        sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]);                                                                         \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce_##rust_type(type *out, type *in, size_t size)                                                                                \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < size)                                                                                                                                            \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = min_##rust_type(in[i], in[i + blockDim.x]);                                                                                              \
        }                                                                                                                                                                     \
        else if (i < size)                                                                                                                                                    \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[i];                                                                                                                                   \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                 \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                           \
    }                                                                                                                                                                         \
    extern "C" __global__ void uncontiguous_reduce_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t size)                           \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < size)                                                                                                                                            \
        {                                                                                                                                                                     \
            long long a_amount = i;                                                                                                                                           \
            long long b_amount = i + blockDim.x;                                                                                                                              \
            long long a_offset = 0;                                                                                                                                           \
            long long b_offset = 0;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                               \
                b_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = min_##rust_type(in[a_offset], in[b_offset]);                                                                                             \
        }                                                                                                                                                                     \
        else if (i < size)                                                                                                                                                    \
        {                                                                                                                                                                     \
            long long a_amount = i;                                                                                                                                           \
            long long a_offset = 0;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = in[a_offset];                                                                                                                            \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                 \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                           \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce2_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t num_blocks_per_row) \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < cols)                                                                                                                                            \
        {                                                                                                                                                                     \
            long long a_offset = 0;                                                                                                                                           \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                       \
            long long b_offset = 0;                                                                                                                                           \
            long long b_amount = i + blockDim.x + blockIdx.y * cols;                                                                                                          \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                               \
                b_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = min_##rust_type(in[a_offset], in[b_offset]);                                                                                             \
        }                                                                                                                                                                     \
        else if (i < cols)                                                                                                                                                    \
        {                                                                                                                                                                     \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                       \
                                                                                                                                                                              \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                     \
                a_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = *in;                                                                                                                                     \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                 \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                         \
    }                                                                                                                                                                         \
                                                                                                                                                                              \
    extern "C" __global__ void contiguous_reduce22_##rust_type(type *out, type *in, size_t cols, size_t num_blocks_per_row)                                                   \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < cols)                                                                                                                                            \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = min_##rust_type(in[i + blockIdx.y * cols], in[i + blockDim.x + blockIdx.y * cols]);                                                      \
        }                                                                                                                                                                     \
        else if (i < cols)                                                                                                                                                    \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                               \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                 \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                         \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce3_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t rows)               \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                            \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                         \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                         \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                               \
        {                                                                                                                                                                     \
            return;                                                                                                                                                           \
        }                                                                                                                                                                     \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                          \
        long long offset = 0;                                                                                                                                                 \
        for (int j = ndim - 1; j >= 0; j--)                                                                                                                                   \
        {                                                                                                                                                                     \
            offset += (idx % shape[j]) * strides[j];                                                                                                                          \
            idx /= shape[j];                                                                                                                                                  \
        }                                                                                                                                                                     \
        sdata_##rust_type[tid] = in[offset];                                                                                                                                  \
        __syncthreads();                                                                                                                                                      \
        if (threadIdx.y == 0)                                                                                                                                                 \
        {                                                                                                                                                                     \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                     \
            {                                                                                                                                                                 \
                sdata_##rust_type[threadIdx.x] = min_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                            \
            }                                                                                                                                                                 \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                \
        }                                                                                                                                                                     \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce33_##rust_type(type *out, type *in, size_t ndim, size_t cols, size_t rows)                                                    \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                            \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                         \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                         \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                               \
        {                                                                                                                                                                     \
            return;                                                                                                                                                           \
        }                                                                                                                                                                     \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                          \
        sdata_##rust_type[tid] = in[idx];                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        if (threadIdx.y == 0)                                                                                                                                                 \
        {                                                                                                                                                                     \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                     \
            {                                                                                                                                                                 \
                sdata_##rust_type[threadIdx.x] = min_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                            \
            }                                                                                                                                                                 \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                \
        }                                                                                                                                                                     \
    }

DEFINE_REDUCE_KERNEL(bool, bool, BOOL_MAX)
DEFINE_REDUCE_KERNEL(i8, char, I8_MAX)
DEFINE_REDUCE_KERNEL(i16, short, I16_MAX)
DEFINE_REDUCE_KERNEL(i32, int, I32_MAX)
DEFINE_REDUCE_KERNEL(i64, long long, I64_MAX)
DEFINE_REDUCE_KERNEL(u8, unsigned char, U8_MAX)
DEFINE_REDUCE_KERNEL(u16, unsigned short, U16_MAX)
DEFINE_REDUCE_KERNEL(u32, unsigned int, U32_MAX)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, U64_MAX)
DEFINE_REDUCE_KERNEL(f32, float, F32_MAX)
DEFINE_REDUCE_KERNEL(f64, double, F64_MAX)
DEFINE_REDUCE_KERNEL(f16, __half, F16_MAX)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, BF16_MAX)
