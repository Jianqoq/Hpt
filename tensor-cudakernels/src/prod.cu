
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#define WRAP 32

#define BOOL_ONE 1
#define I8_ONE 1
#define I16_ONE 1
#define I32_ONE 1
#define I64_ONE 1
#define F32_ONE 1.0f
#define F64_ONE 1.0
#define F16_ONE __half((unsigned short)1)

#define U8_ONE 1
#define U16_ONE 1
#define U32_ONE 1
#define U64_ONE 1

#define prod_bool(a, b) ((bool)((unsigned char)a) * ((unsigned char)b))
#define prod_i8(a, b) (a) * (b)
#define prod_i16(a, b) (a) * (b)
#define prod_i32(a, b) (a) * (b)
#define prod_i64(a, b) (a) * (b)
#define prod_f32(a, b) (a) * (b)
#define prod_f64(a, b) (a) * (b)

#define prod_u8(a, b) (a) * (b)
#define prod_u16(a, b) (a) * (b)
#define prod_u32(a, b) (a) * (b)
#define prod_u64(a, b) (a) * (b)

#define prod_f16(a, b) __float2half(__half2float((a)) * __half2float((b)))

#define atomicMul_bool(a, b)                       \
    acquire_lock(&global_lock);                    \
    (a) = ((bool)((unsigned char)a) * ((unsigned char)b)); \
    release_lock(&global_lock);

#define atomicMul_i8(a, b)      \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_u8(a, b)      \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_i16(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_u16(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_i64(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_u64(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_i32(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_u32(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_f32(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_f64(a, b)     \
    acquire_lock(&global_lock); \
    (a) *= (b);                 \
    release_lock(&global_lock);

#define atomicMul_f16(a, b)     \
    acquire_lock(&global_lock); \
    (a) = prod_f16(a, b);       \
    release_lock(&global_lock);

__device__ int global_lock = 0;

__device__ void acquire_lock(int *lock)
{
    while (atomicCAS(lock, 0, 1) != 0)
    {
    }
}

__device__ void release_lock(int *lock)
{
    atomicExch(lock, 0);
}

#define DEFINE_REDUCE_KERNEL(rust_type, type, INIT_VAL)                                                                                                                       \
    __device__ __forceinline__ void warpReduce_##rust_type(volatile type *sdata_##rust_type, unsigned int tid)                                                                \
    {                                                                                                                                                                         \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]);                                                                       \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]);                                                                       \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]);                                                                        \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]);                                                                        \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]);                                                                        \
        sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]);                                                                        \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce_##rust_type(type *out, type *in, size_t size)                                                                                \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < size)                                                                                                                                            \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = prod_##rust_type(in[i], in[i + blockDim.x]);                                                                                             \
        }                                                                                                                                                                     \
        else if (i < size)                                                                                                                                                    \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[i];                                                                                                                                   \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                           \
    }                                                                                                                                                                         \
    extern "C" __global__ void uncontiguous_reduce_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t size)                           \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < size)                                                                                                                                            \
        {                                                                                                                                                                     \
            long long a_amount = i;                                                                                                                                           \
            long long b_amount = i + blockDim.x;                                                                                                                              \
            long long a_offset = 0;                                                                                                                                           \
            long long b_offset = 0;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                               \
                b_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = prod_##rust_type(in[a_offset], in[b_offset]);                                                                                            \
        }                                                                                                                                                                     \
        else if (i < size)                                                                                                                                                    \
        {                                                                                                                                                                     \
            long long a_amount = i;                                                                                                                                           \
            long long a_offset = 0;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = in[a_offset];                                                                                                                            \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                           \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce2_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t num_blocks_per_row) \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < cols)                                                                                                                                            \
        {                                                                                                                                                                     \
            long long a_offset = 0;                                                                                                                                           \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                       \
            long long b_offset = 0;                                                                                                                                           \
            long long b_amount = i + blockDim.x + blockIdx.y * cols;                                                                                                          \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                               \
                a_amount /= shape[j];                                                                                                                                         \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                               \
                b_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = prod_##rust_type(in[a_offset], in[b_offset]);                                                                                            \
        }                                                                                                                                                                     \
        else if (i < cols)                                                                                                                                                    \
        {                                                                                                                                                                     \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                       \
                                                                                                                                                                              \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                               \
            {                                                                                                                                                                 \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                     \
                a_amount /= shape[j];                                                                                                                                         \
            }                                                                                                                                                                 \
            sdata_##rust_type[tid] = *in;                                                                                                                                     \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                         \
    }                                                                                                                                                                         \
                                                                                                                                                                              \
    extern "C" __global__ void contiguous_reduce22_##rust_type(type *out, type *in, size_t cols, size_t num_blocks_per_row)                                                   \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                       \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (i + blockDim.x < cols)                                                                                                                                            \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = prod_##rust_type(in[i + blockIdx.y * cols], in[i + blockDim.x + blockIdx.y * cols]);                                                     \
        }                                                                                                                                                                     \
        else if (i < cols)                                                                                                                                                    \
        {                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                               \
        }                                                                                                                                                                     \
        __syncthreads();                                                                                                                                                      \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                              \
        {                                                                                                                                                                     \
            if (tid < s)                                                                                                                                                      \
            {                                                                                                                                                                 \
                sdata_##rust_type[tid] = prod_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                \
            }                                                                                                                                                                 \
            __syncthreads();                                                                                                                                                  \
        }                                                                                                                                                                     \
        if (tid < WRAP)                                                                                                                                                       \
        {                                                                                                                                                                     \
            warpReduce_##rust_type(sdata_##rust_type, tid);                                                                                                                   \
        }                                                                                                                                                                     \
        if (tid == 0)                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                         \
    }                                                                                                                                                                         \
    extern "C" __global__ void contiguous_reduce3_##rust_type(type *out, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t rows)               \
    {                                                                                                                                                                         \
        extern __shared__ type sdata_##rust_type[];                                                                                                                           \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                            \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                         \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                         \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                    \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                               \
        {                                                                                                                                                                     \
            return;                                                                                                                                                           \
        }                                                                                                                                                                     \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                          \
        long long offset = 0;                                                                                                                                                 \
        for (int j = ndim - 1; j >= 0; j--)                                                                                                                                   \
        {                                                                                                                                                                     \
            offset += (idx % shape[j]) * strides[j];                                                                                                                          \
            idx /= shape[j];                                                                                                                                                  \
        }                                                                                                                                                                     \
        sdata_##rust_type[tid] = in[offset];                                                                                                                                  \
        __syncthreads();                                                                                                                                                      \
        if (threadIdx.y == 0)                                                                                                                                                 \
        {                                                                                                                                                                     \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                     \
            {                                                                                                                                                                 \
                sdata_##rust_type[threadIdx.x] = prod_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                           \
            }                                                                                                                                                                 \
            atomicMul_##rust_type(out[col_idx], sdata_##rust_type[threadIdx.x]);                                                                                              \
        }                                                                                                                                                                     \
    }

DEFINE_REDUCE_KERNEL(bool, bool, BOOL_ONE)
DEFINE_REDUCE_KERNEL(i8, char, I8_ONE)
DEFINE_REDUCE_KERNEL(i16, short, I16_ONE)
DEFINE_REDUCE_KERNEL(i32, int, I32_ONE)
DEFINE_REDUCE_KERNEL(i64, long long, I64_ONE)
DEFINE_REDUCE_KERNEL(u8, unsigned char, U8_ONE)
DEFINE_REDUCE_KERNEL(u16, unsigned short, U16_ONE)
DEFINE_REDUCE_KERNEL(u32, unsigned int, U32_ONE)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, U64_ONE)
DEFINE_REDUCE_KERNEL(f32, float, F32_ONE)
DEFINE_REDUCE_KERNEL(f64, double, F64_ONE)
DEFINE_REDUCE_KERNEL(f16, __half, F16_ONE)
