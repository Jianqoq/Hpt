
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <limits.h>

#define WRAP 32

#define BOOL_MAX 1
#define I8_MAX SCHAR_MAX
#define I16_MAX SHRT_MAX
#define I32_MAX INT_MAX
#define I64_MAX LLONG_MAX
#define F32_MAX 1.0f / 0.0f
#define F64_MAX 1.0 / 0.0

#define U8_MAX UCHAR_MAX
#define U16_MAX USHRT_MAX
#define U32_MAX UINT_MAX
#define U64_MAX ULONG_MAX
#define F16_MAX __half((unsigned short)31744)
#define BF16_MAX __hip_bfloat16((unsigned short)0x7F80)

#define min_bool(a, b) ((bool)min(((unsigned char)a), ((unsigned char)b)))
#define min_i8(a, b) min((a), (b))
#define min_i16(a, b) min((a), (b))
#define min_i32(a, b) min((a), (b))
#define min_i64(a, b) min((a), (b))
#define min_f32(a, b) min((a), (b))
#define min_f64(a, b) min((a), (b))

#define min_u8(a, b) min((a), (b))
#define min_u16(a, b) min((a), (b))
#define min_u32(a, b) min((a), (b))
#define min_u64(a, b) min((a), (b))
#define min_f16(a, b) __hmin((a), (b))
#define min_bf16(a, b) __hmin((a), (b))

#define eq_bool(a, b) ((a) == (b))
#define eq_i8(a, b) ((a) == (b))
#define eq_i16(a, b) ((a) == (b))
#define eq_i32(a, b) ((a) == (b))
#define eq_i64(a, b) ((a) == (b))
#define eq_u8(a, b) ((a) == (b))
#define eq_u16(a, b) ((a) == (b))
#define eq_u32(a, b) ((a) == (b))
#define eq_u64(a, b) ((a) == (b))
#define eq_f32(a, b) ((a) == (b))
#define eq_f64(a, b) ((a) == (b))
#define eq_f16(a, b) (__heq((a), (b)))
#define eq_bf16(a, b) (__heq((a), (b)))

#define DEFINE_REDUCE_KERNEL(rust_type, type, INIT_VAL)                                                                                                                                           \
    __device__ __forceinline__ void warpReduce_##rust_type(volatile type *sdata_##rust_type, volatile long long *sdata_##rust_type_idx, unsigned int tid)                                         \
    {                                                                                                                                                                                             \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]), sdata_##rust_type[tid + 32]))                                                                    \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 32];                                                                                                                         \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 32];                                                                                                                                 \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]), sdata_##rust_type[tid + 16]))                                                                    \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 16];                                                                                                                         \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 16];                                                                                                                                 \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]), sdata_##rust_type[tid + 8]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 8];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 8];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]), sdata_##rust_type[tid + 4]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 4];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 4];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]), sdata_##rust_type[tid + 2]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 2];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 2];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]), sdata_##rust_type[tid + 1]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 1];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 1];                                                                                                                                  \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, size_t size)                                                                                \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type min_val = min_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                            \
            if (min_val == in[i + blockDim.x])                                                                                                                                                    \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                      \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                   \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                       \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (min_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce2_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t size)                                                           \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type_idx[tid] = I64_MAX;                                                                                                                                                     \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type min_val = min_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                            \
            if (min_val == in[i + blockDim.x])                                                                                                                                                    \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                      \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockDim.x];                                                                                                                             \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                   \
                sdata_##rust_type_idx[tid] = inp_idx[i];                                                                                                                                          \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                       \
            sdata_##rust_type_idx[tid] = inp_idx[i];                                                                                                                                              \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (min_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void uncontiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t size)                           \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            long long a_amount = i;                                                                                                                                                               \
            long long b_amount = i + blockDim.x;                                                                                                                                                  \
            long long a_offset = 0;                                                                                                                                                               \
            long long b_offset = 0;                                                                                                                                                               \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                   \
                b_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            type min_val = min_##rust_type(in[a_offset], in[b_offset]);                                                                                                                           \
            if (min_val == in[b_offset])                                                                                                                                                          \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[b_offset];                                                                                                                                            \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                            \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            long long a_amount = i;                                                                                                                                                               \
            long long a_offset = 0;                                                                                                                                                               \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            type min_val = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                                   \
            if (min_val == sdata_##rust_type[tid + s])                                                                                                                                            \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                              \
                sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                      \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce3_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t num_blocks_per_row) \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        if (i + blockDim.x < cols)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            long long a_offset = 0;                                                                                                                                                               \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                                           \
            long long b_offset = 0;                                                                                                                                                               \
            long long b_amount = i + blockDim.x + blockIdx.y * cols;                                                                                                                              \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                   \
                b_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            type min_val = min_##rust_type(in[a_offset], in[b_offset]);                                                                                                                           \
            if (min_val == in[b_offset])                                                                                                                                                          \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[b_offset];                                                                                                                                            \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                            \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < cols)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                                         \
                a_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            sdata_##rust_type[tid] = *in;                                                                                                                                                         \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (min_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                                             \
            out_idx[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type_idx[0];                                                                                                     \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce33_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t cols, size_t num_blocks_per_row)                               \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = I64_MAX;                                                                                                                                                     \
        if (i + blockDim.x < cols)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type min_val = min_##rust_type(in[i + blockIdx.y * cols], in[i + blockDim.x + blockIdx.y * cols]);                                                                                    \
            if (min_val == in[i + blockDim.x + blockIdx.y * cols])                                                                                                                                \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                                               \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockIdx.y * cols];                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x + blockIdx.y * cols];                                                                                                                  \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockDim.x + blockIdx.y * cols];                                                                                                         \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < cols)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                                                   \
            sdata_##rust_type_idx[tid] = inp_idx[i + blockIdx.y * cols];                                                                                                                          \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (min_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                                             \
            out_idx[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type_idx[0];                                                                                                     \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce4_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t rows)               \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x * blockDim.y];                                                                                              \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                             \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                             \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = row_idx * cols + col_idx;                                                                                                                                    \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                   \
        {                                                                                                                                                                                         \
            return;                                                                                                                                                                               \
        }                                                                                                                                                                                         \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                              \
        long long offset = 0;                                                                                                                                                                     \
        for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                       \
        {                                                                                                                                                                                         \
            offset += (idx % shape[j]) * strides[j];                                                                                                                                              \
            idx /= shape[j];                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        sdata_##rust_type[tid] = in[offset];                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                          \
        if (threadIdx.y == 0)                                                                                                                                                                     \
        {                                                                                                                                                                                         \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                         \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                                                  \
                if (min_val == sdata_##rust_type[s * blockDim.x + threadIdx.x])                                                                                                                   \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[threadIdx.x] = sdata_##rust_type[s * blockDim.x + threadIdx.x];                                                                                             \
                    sdata_##rust_type_idx[threadIdx.x] = sdata_##rust_type_idx[s * blockDim.x + threadIdx.x];                                                                                     \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            out_idx[col_idx + blockIdx.y * cols] = sdata_##rust_type_idx[threadIdx.x];                                                                                                            \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                                    \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce44_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t ndim, size_t cols, size_t rows)                                \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x * blockDim.y];                                                                                              \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                             \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                             \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                   \
        {                                                                                                                                                                                         \
            return;                                                                                                                                                                               \
        }                                                                                                                                                                                         \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                              \
        sdata_##rust_type[tid] = in[idx];                                                                                                                                                         \
        sdata_##rust_type_idx[tid] = inp_idx[idx];                                                                                                                                                \
        __syncthreads();                                                                                                                                                                          \
        if (threadIdx.y == 0)                                                                                                                                                                     \
        {                                                                                                                                                                                         \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                         \
            {                                                                                                                                                                                     \
                type min_val = min_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                                                  \
                if (min_val == sdata_##rust_type[s * blockDim.x + threadIdx.x])                                                                                                                   \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[threadIdx.x] = sdata_##rust_type[s * blockDim.x + threadIdx.x];                                                                                             \
                    sdata_##rust_type_idx[threadIdx.x] = sdata_##rust_type_idx[s * blockDim.x + threadIdx.x];                                                                                     \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            out_idx[col_idx + blockIdx.y * cols] = sdata_##rust_type_idx[threadIdx.x];                                                                                                            \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                                    \
        }                                                                                                                                                                                         \
    }

DEFINE_REDUCE_KERNEL(bool, bool, BOOL_MAX)
DEFINE_REDUCE_KERNEL(i8, char, I8_MAX)
DEFINE_REDUCE_KERNEL(i16, short, I16_MAX)
DEFINE_REDUCE_KERNEL(i32, int, I32_MAX)
DEFINE_REDUCE_KERNEL(i64, long long, I64_MAX)
DEFINE_REDUCE_KERNEL(u8, unsigned char, U8_MAX)
DEFINE_REDUCE_KERNEL(u16, unsigned short, U16_MAX)
DEFINE_REDUCE_KERNEL(u32, unsigned int, U32_MAX)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, U64_MAX)
DEFINE_REDUCE_KERNEL(f32, float, F32_MAX)
DEFINE_REDUCE_KERNEL(f64, double, F64_MAX)
DEFINE_REDUCE_KERNEL(f16, __half, F16_MAX)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, BF16_MAX)
