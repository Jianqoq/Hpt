
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <limits.h>

#define WRAP 32

#define BOOL_MIN 0
#define I8_MIN SCHAR_MIN
#define I16_MIN SHRT_MIN
#define I32_MIN INT_MIN
#define I64_MIN LLONG_MIN
#define F32_MIN -1.0f / 0.0f
#define F64_MIN -1.0 / 0.0

#define U8_MIN 0
#define U16_MIN 0
#define U32_MIN 0
#define U64_MIN 0
#define F16_MIN __half((unsigned short)0xFC00U)
#define BF16_MIN __hip_bfloat16((unsigned short)0xFF80)

#define max_bool(a, b) ((bool)max(((unsigned char)a), ((unsigned char)b)))
#define max_i8(a, b) max((a), (b))
#define max_i16(a, b) max((a), (b))
#define max_i32(a, b) max((a), (b))
#define max_i64(a, b) max((a), (b))
#define max_f32(a, b) max((a), (b))
#define max_f64(a, b) max((a), (b))

#define max_u8(a, b) max((a), (b))
#define max_u16(a, b) max((a), (b))
#define max_u32(a, b) max((a), (b))
#define max_u64(a, b) max((a), (b))
#define max_f16(a, b) __hmax((a), (b))
#define max_bf16(a, b) __hmax((a), (b))

#define eq_bool(a, b) ((a) == (b))
#define eq_i8(a, b) ((a) == (b))
#define eq_i16(a, b) ((a) == (b))
#define eq_i32(a, b) ((a) == (b))
#define eq_i64(a, b) ((a) == (b))
#define eq_u8(a, b) ((a) == (b))
#define eq_u16(a, b) ((a) == (b))
#define eq_u32(a, b) ((a) == (b))
#define eq_u64(a, b) ((a) == (b))
#define eq_f32(a, b) ((a) == (b))
#define eq_f64(a, b) ((a) == (b))
#define eq_f16(a, b) (__heq((a), (b)))
#define eq_bf16(a, b) (__heq((a), (b)))

#define DEFINE_REDUCE_KERNEL(rust_type, type, INIT_VAL)                                                                                                                                           \
    __device__ __forceinline__ void warpReduce_##rust_type(volatile type *sdata_##rust_type, volatile long long *sdata_##rust_type_idx, unsigned int tid)                                         \
    {                                                                                                                                                                                             \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]), sdata_##rust_type[tid + 32]))                                                                    \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 32];                                                                                                                         \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 32];                                                                                                                                 \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]), sdata_##rust_type[tid + 16]))                                                                    \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 16];                                                                                                                         \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 16];                                                                                                                                 \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]), sdata_##rust_type[tid + 8]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 8];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 8];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]), sdata_##rust_type[tid + 4]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 4];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 4];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]), sdata_##rust_type[tid + 2]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 2];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 2];                                                                                                                                  \
        }                                                                                                                                                                                         \
        if (eq_##rust_type(max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]), sdata_##rust_type[tid + 1]))                                                                      \
        {                                                                                                                                                                                         \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 1];                                                                                                                          \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 1];                                                                                                                                  \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, size_t size)                                                                                \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type max_val = max_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                            \
            if (max_val == in[i + blockDim.x])                                                                                                                                                    \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                      \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                   \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                       \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (max_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce2_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t size)                                                           \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type_idx[tid] = I64_MIN;                                                                                                                                                     \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type max_val = max_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                            \
            if (max_val == in[i + blockDim.x])                                                                                                                                                    \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                      \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockDim.x];                                                                                                                             \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                   \
                sdata_##rust_type_idx[tid] = inp_idx[i];                                                                                                                                          \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                       \
            sdata_##rust_type_idx[tid] = inp_idx[i];                                                                                                                                              \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (max_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void uncontiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t size)                           \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        if (i + blockDim.x < size)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            long long a_amount = i;                                                                                                                                                               \
            long long b_amount = i + blockDim.x;                                                                                                                                                  \
            long long a_offset = 0;                                                                                                                                                               \
            long long b_offset = 0;                                                                                                                                                               \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                   \
                b_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            type max_val = max_##rust_type(in[a_offset], in[b_offset]);                                                                                                                           \
            if (max_val == in[b_offset])                                                                                                                                                          \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[b_offset];                                                                                                                                            \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                            \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < size)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            long long a_amount = i;                                                                                                                                                               \
            long long a_offset = 0;                                                                                                                                                               \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            type max_val = max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                                   \
            if (max_val == sdata_##rust_type[tid + s])                                                                                                                                            \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                              \
                sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                      \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                               \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                       \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce3_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t num_blocks_per_row) \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = i;                                                                                                                                                           \
        if (i + blockDim.x < cols)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            long long a_offset = 0;                                                                                                                                                               \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                                           \
            long long b_offset = 0;                                                                                                                                                               \
            long long b_amount = i + blockDim.x + blockIdx.y * cols;                                                                                                                              \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                             \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                   \
                b_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            type max_val = max_##rust_type(in[a_offset], in[b_offset]);                                                                                                                           \
            if (max_val == in[b_offset])                                                                                                                                                          \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[b_offset];                                                                                                                                            \
                sdata_##rust_type_idx[tid] = i + blockDim.x;                                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                            \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < cols)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            long long a_amount = i + blockIdx.y * cols;                                                                                                                                           \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                   \
            {                                                                                                                                                                                     \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                                         \
                a_amount /= shape[j];                                                                                                                                                             \
            }                                                                                                                                                                                     \
            sdata_##rust_type[tid] = *in;                                                                                                                                                         \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (max_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                                             \
            out_idx[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type_idx[0];                                                                                                     \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce33_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t cols, size_t num_blocks_per_row)                               \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                           \
        unsigned int tid = threadIdx.x;                                                                                                                                                           \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                               \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = I64_MIN;                                                                                                                                                     \
        if (i + blockDim.x < cols)                                                                                                                                                                \
        {                                                                                                                                                                                         \
            type max_val = max_##rust_type(in[i + blockIdx.y * cols], in[i + blockDim.x + blockIdx.y * cols]);                                                                                    \
            if (max_val == in[i + blockDim.x + blockIdx.y * cols])                                                                                                                                \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                                               \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockIdx.y * cols];                                                                                                                      \
            }                                                                                                                                                                                     \
            else                                                                                                                                                                                  \
            {                                                                                                                                                                                     \
                sdata_##rust_type[tid] = in[i + blockDim.x + blockIdx.y * cols];                                                                                                                  \
                sdata_##rust_type_idx[tid] = inp_idx[i + blockDim.x + blockIdx.y * cols];                                                                                                         \
            }                                                                                                                                                                                     \
        }                                                                                                                                                                                         \
        else if (i < cols)                                                                                                                                                                        \
        {                                                                                                                                                                                         \
            sdata_##rust_type[tid] = in[i + blockIdx.y * cols];                                                                                                                                   \
            sdata_##rust_type_idx[tid] = inp_idx[i + blockIdx.y * cols];                                                                                                                          \
        }                                                                                                                                                                                         \
        __syncthreads();                                                                                                                                                                          \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                  \
        {                                                                                                                                                                                         \
            if (tid < s)                                                                                                                                                                          \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]);                                                                                               \
                if (max_val == sdata_##rust_type[tid + s])                                                                                                                                        \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                          \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                  \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            __syncthreads();                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        if (tid < WRAP)                                                                                                                                                                           \
        {                                                                                                                                                                                         \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                \
        }                                                                                                                                                                                         \
        if (tid == 0)                                                                                                                                                                             \
        {                                                                                                                                                                                         \
            out[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type[0];                                                                                                             \
            out_idx[blockIdx.x + blockIdx.y * num_blocks_per_row] = sdata_##rust_type_idx[0];                                                                                                     \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce4_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t rows)               \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x * blockDim.y];                                                                                              \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                             \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                             \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        sdata_##rust_type_idx[tid] = row_idx * cols + col_idx;                                                                                                                                    \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                   \
        {                                                                                                                                                                                         \
            return;                                                                                                                                                                               \
        }                                                                                                                                                                                         \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                              \
        long long offset = 0;                                                                                                                                                                     \
        for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                       \
        {                                                                                                                                                                                         \
            offset += (idx % shape[j]) * strides[j];                                                                                                                                              \
            idx /= shape[j];                                                                                                                                                                      \
        }                                                                                                                                                                                         \
        sdata_##rust_type[tid] = in[offset];                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                          \
        if (threadIdx.y == 0)                                                                                                                                                                     \
        {                                                                                                                                                                                         \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                         \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                                                  \
                if (max_val == sdata_##rust_type[s * blockDim.x + threadIdx.x])                                                                                                                   \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[threadIdx.x] = sdata_##rust_type[s * blockDim.x + threadIdx.x];                                                                                             \
                    sdata_##rust_type_idx[threadIdx.x] = sdata_##rust_type_idx[s * blockDim.x + threadIdx.x];                                                                                     \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            out_idx[col_idx + blockIdx.y * cols] = sdata_##rust_type_idx[threadIdx.x];                                                                                                            \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                                    \
        }                                                                                                                                                                                         \
    }                                                                                                                                                                                             \
    extern "C" __global__ void contiguous_reduce44_##rust_type(type *out, long long *out_idx, type *in, long long *inp_idx, size_t ndim, size_t cols, size_t rows)                                \
    {                                                                                                                                                                                             \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                               \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x * blockDim.y];                                                                                              \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                             \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                             \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                        \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                   \
        {                                                                                                                                                                                         \
            return;                                                                                                                                                                               \
        }                                                                                                                                                                                         \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                              \
        sdata_##rust_type[tid] = in[idx];                                                                                                                                                         \
        sdata_##rust_type_idx[tid] = inp_idx[idx];                                                                                                                                                \
        __syncthreads();                                                                                                                                                                          \
        if (threadIdx.y == 0)                                                                                                                                                                     \
        {                                                                                                                                                                                         \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                         \
            {                                                                                                                                                                                     \
                type max_val = max_##rust_type(sdata_##rust_type[threadIdx.x], sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                                                  \
                if (max_val == sdata_##rust_type[s * blockDim.x + threadIdx.x])                                                                                                                   \
                {                                                                                                                                                                                 \
                    sdata_##rust_type[threadIdx.x] = sdata_##rust_type[s * blockDim.x + threadIdx.x];                                                                                             \
                    sdata_##rust_type_idx[threadIdx.x] = sdata_##rust_type_idx[s * blockDim.x + threadIdx.x];                                                                                     \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            out_idx[col_idx + blockIdx.y * cols] = sdata_##rust_type_idx[threadIdx.x];                                                                                                            \
            out[col_idx + blockIdx.y * cols] = sdata_##rust_type[threadIdx.x];                                                                                                                    \
        }                                                                                                                                                                                         \
    }

DEFINE_REDUCE_KERNEL(bool, bool, BOOL_MIN)
DEFINE_REDUCE_KERNEL(i8, char, I8_MIN)
DEFINE_REDUCE_KERNEL(i16, short, I16_MIN)
DEFINE_REDUCE_KERNEL(i32, int, I32_MIN)
DEFINE_REDUCE_KERNEL(i64, long long, I64_MIN)
DEFINE_REDUCE_KERNEL(u8, unsigned char, U8_MIN)
DEFINE_REDUCE_KERNEL(u16, unsigned short, U16_MIN)
DEFINE_REDUCE_KERNEL(u32, unsigned int, U32_MIN)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, U64_MIN)
DEFINE_REDUCE_KERNEL(f32, float, F32_MIN)
DEFINE_REDUCE_KERNEL(f64, double, F64_MIN)
DEFINE_REDUCE_KERNEL(f16, half, F16_MIN)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, BF16_MIN)
