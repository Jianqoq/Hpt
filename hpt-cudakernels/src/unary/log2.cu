#include "hip/hip_runtime.h"
#include "unary_template.cuh"
#include "../utils/type_alias.cuh"
#include "../utils/promotion/promotes.cuh"
#include "../utils/check_type.cuh"

template <typename Input>
struct Log2
{
    using Output = typename FloatOutUnaryPromote<Input>::Output;
    __device__ __forceinline__ Output operator()(Input a) const
    {
        CHECK_FLOAT_TYPE(Output);
        if constexpr (std::is_same_v<Input, f16> || std::is_same_v<Input, bf16>)
        {
            return cast<f32, Output>(log2(cast<Input, f32>(a)));
        }
        else
        {
            return log2(cast<Input, Output>(a));
        }
    }
};

DEFINE_UNARY_KERNEL(log2_f16, f16, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_bf16, bf16, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_f32, f32, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_f64, f64, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_bool, bool, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_i8, i8, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_i16, i16, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_i32, i32, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_i64, i64, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_u8, u8, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_u16, u16, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_u32, u32, FloatOutUnaryPromote, Log2);
DEFINE_UNARY_KERNEL(log2_u64, u64, FloatOutUnaryPromote, Log2);
