#include "hip/hip_runtime.h"
#include "unary_template.cuh"
#include "../utils/type_alias.cuh"
#include "../utils/promotion/promotes.cuh"
#include "../utils/check_type.cuh"

template <typename Input>
struct Log10
{
    using Output = typename FloatOutUnaryPromote<Input>::Output;
    __device__ __forceinline__ Output operator()(Input a) const
    {
        CHECK_FLOAT_TYPE(Output);
        if constexpr (std::is_same_v<Input, f16> || std::is_same_v<Input, bf16>)
        {
            return cast<f32, Output>(log10(cast<Input, f32>(a)));
        }
        else
        {
            return log10(cast<Input, Output>(a));
        }
    }
};

DEFINE_UNARY_KERNEL(log10_f16, f16, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_bf16, bf16, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_f32, f32, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_f64, f64, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_bool, bool, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_i8, i8, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_i16, i16, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_i32, i32, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_i64, i64, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_u8, u8, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_u16, u16, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_u32, u32, FloatOutUnaryPromote, Log10);
DEFINE_UNARY_KERNEL(log10_u64, u64, FloatOutUnaryPromote, Log10);
