#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

void gen_rand_data(float *data, int n);

template <int kTileM, int kTileN, int kTileK, typename TiledMMA>
__device__ void gemm_simple(float *Cptr, const float *Aptr, const float *Bptr, int m, int n, int k)
{

    using namespace cute;

    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

    int ix = blockIdx.x;
    int iy = blockIdx.y;

    Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
    Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
    Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix));
    //  gA(kTileM, kTileK, num_tile_k)
    //  gB(kTileN, kTileK, num_tile_k)
    //  gC(kTileM, kTileN)

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tAgA = thr_mma.partition_A(gA); // (MMA, MMA_M, MMA_K, num_tile_k)
    auto tBgB = thr_mma.partition_B(gB); // (MMA, MMA_N, MMA_K, num_tile_k)
    auto tCgC = thr_mma.partition_C(gC); // (MMA, MMA_M, MMA_N)

    auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0)); // (MMA, MMA_M, MMA_K)
    auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0)); // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC(_, _));    // (MMA, MMA_M, MMA_N)

    clear(tCrC);

    int num_tile_k = size<2>(gA);
#pragma unroll 1
    for (int itile = 0; itile < num_tile_k; ++itile)
    {
        cute::copy(tAgA(_, _, _, itile), tArA);
        cute::copy(tBgB(_, _, _, itile), tBrB);

        cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }

    cute::copy(tCrC, tCgC);
}

extern "C" __global__ void gemm_simple_kernel(float *Cptr, const float *Aptr, const float *Bptr, int m, int n, int k)
{
    using namespace cute;
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    using MMA = decltype(make_tiled_mma(mma_atom{},
                                        make_layout(Shape<_2, _2, _1>{}),
                                        make_layout(Shape<_1, _2, _1>{})));
    gemm_simple<16, 16, 16, MMA>(Cptr, Aptr, Bptr, m, n, k);
}
