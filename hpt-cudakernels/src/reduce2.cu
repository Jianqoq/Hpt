/*
reduce2 is similar to reduce.cu, reduce2 uses 2 level op
*/


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#define WRAP 32

#define DEFINE_REDUCE_KERNEL(rust_type, in_type, out_type, INIT_VAL, METHOD)                                                                                                                                                \
    __device__ __forceinline__ void warpReduce_##METHOD##_##rust_type(volatile out_type *METHOD##sdata_##rust_type, unsigned int tid)                                                                                       \
    {                                                                                                                                                                                                                       \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP]);                                                                                      \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP / 2]);                                                                                  \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP / 4]);                                                                                  \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP / 8]);                                                                                  \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP / 16]);                                                                                 \
        METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + WRAP / 32]);                                                                                 \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void contiguous_##METHOD##_##rust_type(out_type *out, in_type *in, size_t size)                                                                                                                   \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (i + blockDim.x < size)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = METHOD##1_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                              \
        }                                                                                                                                                                                                                   \
        else if (i < size)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = METHOD##1_unary_##rust_type(in[i]);                                                                                                                                            \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x] = METHOD##sdata_##rust_type[0];                                                                                                                                                                 \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void contiguous_cumulate_##METHOD##_##rust_type(out_type *out, out_type *in, size_t size)                                                                                                         \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (i + blockDim.x < size)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(in[i], in[i + blockDim.x]);                                                                                                                              \
        }                                                                                                                                                                                                                   \
        else if (i < size)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = in[i];                                                                                                                                                                         \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x] = METHOD##sdata_##rust_type[0];                                                                                                                                                                 \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void uncontiguous_##METHOD##_##rust_type(out_type *out, in_type *in, long long *shape, long long *strides, size_t ndim, size_t size)                                                              \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (i + blockDim.x < size)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            long long a_amount = i;                                                                                                                                                                                         \
            long long b_amount = i + blockDim.x;                                                                                                                                                                            \
            long long a_offset = 0;                                                                                                                                                                                         \
            long long b_offset = 0;                                                                                                                                                                                         \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                             \
                a_amount /= shape[j];                                                                                                                                                                                       \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                                             \
                b_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = METHOD##1_##rust_type(in[a_offset], in[b_offset]);                                                                                                                             \
        }                                                                                                                                                                                                                   \
        else if (i < size)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            long long a_amount = i;                                                                                                                                                                                         \
            long long a_offset = 0;                                                                                                                                                                                         \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                             \
                a_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = METHOD##1_unary_##rust_type(in[a_offset]);                                                                                                                                     \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x] = METHOD##sdata_##rust_type[0];                                                                                                                                                                 \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void uncontiguous_cumulate_##METHOD##_##rust_type(out_type *out, out_type *in, long long *shape, long long *strides, size_t ndim, size_t size)                                                    \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (i + blockDim.x < size)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            long long a_amount = i;                                                                                                                                                                                         \
            long long b_amount = i + blockDim.x;                                                                                                                                                                            \
            long long a_offset = 0;                                                                                                                                                                                         \
            long long b_offset = 0;                                                                                                                                                                                         \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                             \
                a_amount /= shape[j];                                                                                                                                                                                       \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                                             \
                b_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(in[a_offset], in[b_offset]);                                                                                                                             \
        }                                                                                                                                                                                                                   \
        else if (i < size)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            long long a_amount = i;                                                                                                                                                                                         \
            long long a_offset = 0;                                                                                                                                                                                         \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                             \
                a_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                                  \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x] = METHOD##sdata_##rust_type[0];                                                                                                                                                                 \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void contiguous_##METHOD##2_##rust_type(out_type *out, in_type *in, long long *shape, long long *strides, size_t ndim, size_t start_row_idx, size_t cols, size_t rows, size_t num_blocks_per_row) \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (blockIdx.y + start_row_idx >= rows)                                                                                                                                                                             \
        {                                                                                                                                                                                                                   \
            return;                                                                                                                                                                                                         \
        }                                                                                                                                                                                                                   \
        if (i + blockDim.x < cols)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            long long a_offset = 0;                                                                                                                                                                                         \
            long long a_amount = i + (blockIdx.y + start_row_idx) * cols;                                                                                                                                                   \
            long long b_offset = 0;                                                                                                                                                                                         \
            long long b_amount = i + blockDim.x + (blockIdx.y + start_row_idx) * cols;                                                                                                                                      \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                             \
                a_amount /= shape[j];                                                                                                                                                                                       \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                                             \
                b_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = METHOD##1_##rust_type(in[a_offset], in[b_offset]);                                                                                                                             \
        }                                                                                                                                                                                                                   \
        else if (i < cols)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            long long a_amount = i + (blockIdx.y + start_row_idx) * cols;                                                                                                                                                   \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                             \
            {                                                                                                                                                                                                               \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                                                                   \
                a_amount /= shape[j];                                                                                                                                                                                       \
            }                                                                                                                                                                                                               \
            METHOD##sdata_##rust_type[tid] = METHOD##1_unary_##rust_type(*in);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x + (blockIdx.y + start_row_idx) * num_blocks_per_row] = METHOD##sdata_##rust_type[0];                                                                                                             \
    }                                                                                                                                                                                                                       \
                                                                                                                                                                                                                            \
    extern "C" __global__ void contiguous_##METHOD##22_##rust_type(out_type *out, out_type *in, size_t start_row_idx, size_t cols, size_t rows, size_t num_blocks_per_row)                                                  \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                     \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                         \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (blockIdx.y + start_row_idx >= rows)                                                                                                                                                                             \
        {                                                                                                                                                                                                                   \
            return;                                                                                                                                                                                                         \
        }                                                                                                                                                                                                                   \
        if (i + blockDim.x < cols)                                                                                                                                                                                          \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(in[i + (blockIdx.y + start_row_idx) * cols], in[i + blockDim.x + (blockIdx.y + start_row_idx) * cols]);                                                  \
        }                                                                                                                                                                                                                   \
        else if (i < cols)                                                                                                                                                                                                  \
        {                                                                                                                                                                                                                   \
            METHOD##sdata_##rust_type[tid] = in[i + (blockIdx.y + start_row_idx) * cols];                                                                                                                                   \
        }                                                                                                                                                                                                                   \
        __syncthreads();                                                                                                                                                                                                    \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                            \
        {                                                                                                                                                                                                                   \
            if (tid < s)                                                                                                                                                                                                    \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[tid] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[tid], METHOD##sdata_##rust_type[tid + s]);                                                                                 \
            }                                                                                                                                                                                                               \
            __syncthreads();                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        if (tid < WRAP)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                   \
            warpReduce_##METHOD##_##rust_type(METHOD##sdata_##rust_type, tid);                                                                                                                                              \
        }                                                                                                                                                                                                                   \
        if (tid == 0)                                                                                                                                                                                                       \
            out[blockIdx.x + (blockIdx.y + start_row_idx) * num_blocks_per_row] = METHOD##sdata_##rust_type[0];                                                                                                             \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void contiguous_##METHOD##3_##rust_type(out_type *out, in_type *in, long long *shape, long long *strides, size_t ndim, size_t cols, size_t rows)                                                  \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                                          \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                       \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                                                       \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                                             \
        {                                                                                                                                                                                                                   \
            return;                                                                                                                                                                                                         \
        }                                                                                                                                                                                                                   \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                                                        \
        long long offset = 0;                                                                                                                                                                                               \
        for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                                 \
        {                                                                                                                                                                                                                   \
            offset += (idx % shape[j]) * strides[j];                                                                                                                                                                        \
            idx /= shape[j];                                                                                                                                                                                                \
        }                                                                                                                                                                                                                   \
        METHOD##sdata_##rust_type[tid] = METHOD##1_unary_##rust_type(in[offset]);                                                                                                                                           \
        __syncthreads();                                                                                                                                                                                                    \
        if (threadIdx.y == 0)                                                                                                                                                                                               \
        {                                                                                                                                                                                                                   \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                                                   \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[threadIdx.x] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[threadIdx.x], METHOD##sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                            \
            }                                                                                                                                                                                                               \
            out[col_idx + blockIdx.y * cols] = METHOD##sdata_##rust_type[threadIdx.x];                                                                                                                                      \
        }                                                                                                                                                                                                                   \
    }                                                                                                                                                                                                                       \
    extern "C" __global__ void contiguous_##METHOD##33_##rust_type(out_type *out, out_type *in, size_t ndim, size_t cols, size_t rows)                                                                                      \
    {                                                                                                                                                                                                                       \
        extern __shared__ out_type METHOD##sdata_##rust_type[];                                                                                                                                                             \
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;                                                                                                                                                          \
        unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                       \
        unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;                                                                                                                                                       \
        METHOD##sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                          \
        if (col_idx >= cols || row_idx >= rows)                                                                                                                                                                             \
        {                                                                                                                                                                                                                   \
            return;                                                                                                                                                                                                         \
        }                                                                                                                                                                                                                   \
        unsigned int idx = row_idx * cols + col_idx;                                                                                                                                                                        \
        METHOD##sdata_##rust_type[tid] = in[idx];                                                                                                                                                                           \
        __syncthreads();                                                                                                                                                                                                    \
        if (threadIdx.y == 0)                                                                                                                                                                                               \
        {                                                                                                                                                                                                                   \
            for (unsigned int s = 1; s < blockDim.y; s++)                                                                                                                                                                   \
            {                                                                                                                                                                                                               \
                METHOD##sdata_##rust_type[threadIdx.x] = METHOD##2_##rust_type(METHOD##sdata_##rust_type[threadIdx.x], METHOD##sdata_##rust_type[s * blockDim.x + threadIdx.x]);                                            \
            }                                                                                                                                                                                                               \
            out[col_idx + blockIdx.y * cols] = METHOD##sdata_##rust_type[threadIdx.x];                                                                                                                                      \
        }                                                                                                                                                                                                                   \
    }

// sum_square
#define sum_square1_bool(a, b) ((bool)(((unsigned int)(a)) * ((unsigned int)(a)) + ((unsigned int)(b)) * ((unsigned int)(b))))
#define sum_square1_i8(a, b) (a) * (a) + (b) * (b)
#define sum_square1_i16(a, b) (a) * (a) + (b) * (b)
#define sum_square1_i32(a, b) (a) * (a) + (b) * (b)
#define sum_square1_i64(a, b) (a) * (a) + (b) * (b)
#define sum_square1_f32(a, b) (a) * (a) + (b) * (b)
#define sum_square1_f64(a, b) (a) * (a) + (b) * (b)

#define sum_square1_u8(a, b) (a) * (a) + (b) * (b)
#define sum_square1_u16(a, b) (a) * (a) + (b) * (b)
#define sum_square1_u32(a, b) (a) * (a) + (b) * (b)
#define sum_square1_u64(a, b) (a) * (a) + (b) * (b)

#define sum_square1_f16(a, b) (a) * (a) + (b) * (b)
#define sum_square1_bf16(a, b) (a) * (a) + (b) * (b)

#define sum_square1_unary_bool(a) ((bool)(((unsigned int)(a)) * ((unsigned int)(a))))
#define sum_square1_unary_i8(a) (a) * (a)
#define sum_square1_unary_i16(a) (a) * (a)
#define sum_square1_unary_i32(a) (a) * (a)
#define sum_square1_unary_i64(a) (a) * (a)
#define sum_square1_unary_f32(a) (a) * (a)
#define sum_square1_unary_f64(a) (a) * (a)

#define sum_square1_unary_u8(a) (a) * (a)
#define sum_square1_unary_u16(a) (a) * (a)
#define sum_square1_unary_u32(a) (a) * (a)
#define sum_square1_unary_u64(a) (a) * (a)

#define sum_square1_unary_f16(a) (a) * (a)
#define sum_square1_unary_bf16(a) (a) * (a)

#define sum_square2_bool(a, b) ((bool)(((unsigned int)(a)) + ((unsigned int)(b))))
#define sum_square2_i8(a, b) (a) + (b)
#define sum_square2_i16(a, b) (a) + (b)
#define sum_square2_i32(a, b) (a) + (b)
#define sum_square2_i64(a, b) (a) + (b)
#define sum_square2_f32(a, b) (a) + (b)
#define sum_square2_f64(a, b) (a) + (b)

#define sum_square2_u8(a, b) (a) + (b)
#define sum_square2_u16(a, b) (a) + (b)
#define sum_square2_u32(a, b) (a) + (b)
#define sum_square2_u64(a, b) (a) + (b)

#define sum_square2_f16(a, b) __hadd((a), (b))
#define sum_square2_bf16(a, b) __hadd((a), (b))

DEFINE_REDUCE_KERNEL(bool, bool, bool, 0, sum_square)
DEFINE_REDUCE_KERNEL(i8, char, char, 0, sum_square)
DEFINE_REDUCE_KERNEL(i16, short, short, 0, sum_square)
DEFINE_REDUCE_KERNEL(i32, int, int, 0, sum_square)
DEFINE_REDUCE_KERNEL(i64, long long, long long, 0, sum_square)

DEFINE_REDUCE_KERNEL(u8, unsigned char, unsigned char, 0, sum_square)
DEFINE_REDUCE_KERNEL(u16, unsigned short, unsigned short, 0, sum_square)
DEFINE_REDUCE_KERNEL(u32, unsigned int, unsigned int, 0, sum_square)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, unsigned long long, 0, sum_square)

DEFINE_REDUCE_KERNEL(f32, float, float, 0.0f, sum_square)
DEFINE_REDUCE_KERNEL(f64, double, double, 0.0, sum_square)

DEFINE_REDUCE_KERNEL(f16, __half, __half, 0, sum_square)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, __hip_bfloat16, 0, sum_square)

// reducel3
#define reducel31_bool(a, b) __float2half_rn(powf((float)(a), 3.0f) + powf((float)(b), 3.0f))
#define reducel31_i8(a, b) __float2half_rn(powf((float)abs(a), 3.0f) + powf((float)abs(b), 3.0f))
#define reducel31_i16(a, b) __float2half_rn(powf((float)abs(a), 3.0f) + powf((float)abs(b), 3.0f))
#define reducel31_i32(a, b) powf((float)abs(a), 3.0f) + powf((float)abs(b), 3.0f)
#define reducel31_i64(a, b) pow((double)abs(a), 3.0) + pow((double)abs(b), 3.0)
#define reducel31_f32(a, b) powf(abs(a), 3.0f) + powf(abs(b), 3.0f)
#define reducel31_f64(a, b) pow(abs(a), 3.0) + pow(abs(b), 3.0)

#define reducel31_u8(a, b) __float2half_rn(powf((float)(a), 3.0f) + powf((float)(b), 3.0f))
#define reducel31_u16(a, b) __float2half_rn(powf((float)(a), 3.0f) + powf((float)(b), 3.0f))
#define reducel31_u32(a, b) powf((float)(a), 3.0f) + powf((float)(b), 3.0f)
#define reducel31_u64(a, b) pow((double)(a), 3.0) + pow((double)(b), 3.0)

#define reducel31_f16(a, b) __float2half_rn(powf(abs(__half2float(a)), 3.0f) + powf(abs(__half2float(b)), 3.0f))
#define reducel31_bf16(a, b) __float2bfloat16_rn(powf(abs(__bfloat162float(a)), 3.0f) + powf(abs(__bfloat162float(b)), 3.0f))

#define reducel31_unary_bool(a) __float2half_rn(powf((float)(a), 3.0f))
#define reducel31_unary_i8(a) __float2half_rn(powf((float)abs(a), 3.0f))
#define reducel31_unary_i16(a) __float2half_rn(powf((float)abs(a), 3.0f))
#define reducel31_unary_i32(a) powf((float)abs(a), 3.0f)
#define reducel31_unary_i64(a) pow((double)abs(a), 3.0)
#define reducel31_unary_f32(a) powf(abs(a), 3.0f)
#define reducel31_unary_f64(a) pow(abs(a), 3.0)

#define reducel31_unary_u8(a) __float2half_rn(powf((float)(a), 3.0f))
#define reducel31_unary_u16(a) __float2half_rn(powf((float)(a), 3.0f))
#define reducel31_unary_u32(a) powf((float)(a), 3.0f)
#define reducel31_unary_u64(a) pow((double)(a), 3.0)

#define reducel31_unary_f16(a) __float2half_rn(powf(abs(__half2float(a)), 3.0f))
#define reducel31_unary_bf16(a) __float2bfloat16_rn(powf(abs(__bfloat162float(a)), 3.0f))

#define reducel32_bool(a, b) __hadd((a), (b))
#define reducel32_i8(a, b) __hadd((a), (b))
#define reducel32_i16(a, b) __hadd((a), (b))
#define reducel32_i32(a, b) (a) + (b)
#define reducel32_i64(a, b) (a) + (b)
#define reducel32_f32(a, b) (a) + (b)
#define reducel32_f64(a, b) (a) + (b)

#define reducel32_u8(a, b) __hadd((a), (b))
#define reducel32_u16(a, b) __hadd((a), (b))
#define reducel32_u32(a, b) (a) + (b)
#define reducel32_u64(a, b) (a) + (b)

#define reducel32_f16(a, b) __hadd((a), (b))
#define reducel32_bf16(a, b) __hadd((a), (b))

DEFINE_REDUCE_KERNEL(bool, bool, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(i8, char, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(i16, short, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(i32, int, float, 0, reducel3)
DEFINE_REDUCE_KERNEL(i64, long long, double, 0, reducel3)

DEFINE_REDUCE_KERNEL(u8, unsigned char, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(u16, unsigned short, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(u32, unsigned int, float, 0, reducel3)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, double, 0, reducel3)

DEFINE_REDUCE_KERNEL(f32, float, float, 0.0f, reducel3)
DEFINE_REDUCE_KERNEL(f64, double, double, 0.0, reducel3)

DEFINE_REDUCE_KERNEL(f16, __half, __half, 0, reducel3)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, __hip_bfloat16, 0, reducel3)

// mean
#define mean1_bool(a, b) __float2half_rn((float)(a) + (float)(b))
#define mean1_i8(a, b) __float2half_rn((float)(a + b))
#define mean1_i16(a, b) __float2half_rn((float)(a + b))
#define mean1_i32(a, b) (float)(a + b)
#define mean1_i64(a, b) (double)(a + b)
#define mean1_f32(a, b) (a) + (b)
#define mean1_f64(a, b) (a) + (b)

#define mean1_u8(a, b) __float2half_rn((float)(a + b))
#define mean1_u16(a, b) __float2half_rn((float)(a + b))
#define mean1_u32(a, b) (float)(a + b)
#define mean1_u64(a, b) (double)(a + b)

#define mean1_f16(a, b) __hadd((a), (b))
#define mean1_bf16(a, b) __hadd((a), (b))

#define mean1_unary_bool(a) __float2half_rn((float)(a))
#define mean1_unary_i8(a) __float2half_rn((float)(a))
#define mean1_unary_i16(a) __float2half_rn((float)(a))
#define mean1_unary_i32(a) (float)(a)
#define mean1_unary_i64(a) (double)(a)
#define mean1_unary_f32(a) (a)
#define mean1_unary_f64(a) (a)

#define mean1_unary_u8(a) __float2half_rn((float)(a))
#define mean1_unary_u16(a) __float2half_rn((float)(a))
#define mean1_unary_u32(a) (float)(a)
#define mean1_unary_u64(a) (double)(a)

#define mean1_unary_f16(a) (a)
#define mean1_unary_bf16(a) (a)

#define mean2_bool(a, b) __hadd((a), (b))
#define mean2_i8(a, b) __hadd((a), (b))
#define mean2_i16(a, b) __hadd((a), (b))
#define mean2_i32(a, b) (a) + (b)
#define mean2_i64(a, b) (a) + (b)
#define mean2_f32(a, b) (a) + (b)
#define mean2_f64(a, b) (a) + (b)

#define mean2_u8(a, b) __hadd((a), (b))
#define mean2_u16(a, b) __hadd((a), (b))
#define mean2_u32(a, b) (a) + (b)
#define mean2_u64(a, b) (a) + (b)

#define mean2_f16(a, b) __hadd((a), (b))
#define mean2_bf16(a, b) __hadd((a), (b))

DEFINE_REDUCE_KERNEL(bool, bool, __half, 0, mean)
DEFINE_REDUCE_KERNEL(i8, char, __half, 0, mean)
DEFINE_REDUCE_KERNEL(i16, short, __half, 0, mean)
DEFINE_REDUCE_KERNEL(i32, int, float, 0, mean)
DEFINE_REDUCE_KERNEL(i64, long long, double, 0, mean)

DEFINE_REDUCE_KERNEL(u8, unsigned char, __half, 0, mean)
DEFINE_REDUCE_KERNEL(u16, unsigned short, __half, 0, mean)
DEFINE_REDUCE_KERNEL(u32, unsigned int, float, 0, mean)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, double, 0, mean)

DEFINE_REDUCE_KERNEL(f32, float, float, 0.0f, mean)
DEFINE_REDUCE_KERNEL(f64, double, double, 0.0, mean)

DEFINE_REDUCE_KERNEL(f16, __half, __half, 0, mean)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, __hip_bfloat16, 0, mean)

// any
#define any1_bool(a, b) (a) || (b)
#define any1_i8(a, b) (a) || (b)
#define any1_i16(a, b) (a) || (b)
#define any1_i32(a, b) (a) || (b)
#define any1_i64(a, b) (a) || (b)
#define any1_f32(a, b) (a) || (b)
#define any1_f64(a, b) (a) || (b)

#define any1_u8(a, b) (a) || (b)
#define any1_u16(a, b) (a) || (b)
#define any1_u32(a, b) (a) || (b)
#define any1_u64(a, b) (a) || (b)

#define any1_f16(a, b) (a) || (b)
#define any1_bf16(a, b) (a) || (b)

#define any1_unary_bool(a) (a)
#define any1_unary_i8(a) ((bool)a)
#define any1_unary_i16(a) ((bool)a)
#define any1_unary_i32(a) ((bool)a)
#define any1_unary_i64(a) ((bool)a)
#define any1_unary_f32(a) ((bool)a)
#define any1_unary_f64(a) ((bool)a)

#define any1_unary_u8(a) ((bool)a)
#define any1_unary_u16(a) ((bool)a)
#define any1_unary_u32(a) ((bool)a)
#define any1_unary_u64(a) ((bool)a)

#define any1_unary_f16(a) ((bool)__half2float(a))
#define any1_unary_bf16(a) ((bool)__bfloat162float(a))

#define any2_bool(a, b) (a) || (b)
#define any2_i8(a, b) (a) || (b)
#define any2_i16(a, b) (a) || (b)
#define any2_i32(a, b) (a) || (b)
#define any2_i64(a, b) (a) || (b)
#define any2_f32(a, b) (a) || (b)
#define any2_f64(a, b) (a) || (b)

#define any2_u8(a, b) (a) || (b)
#define any2_u16(a, b) (a) || (b)
#define any2_u32(a, b) (a) || (b)
#define any2_u64(a, b) (a) || (b)

#define any2_f16(a, b) (a) || (b)
#define any2_bf16(a, b) (a) || (b)

DEFINE_REDUCE_KERNEL(bool, bool, bool, false, any)
DEFINE_REDUCE_KERNEL(i8, char, bool, false, any)
DEFINE_REDUCE_KERNEL(i16, short, bool, false, any)
DEFINE_REDUCE_KERNEL(i32, int, bool, false, any)
DEFINE_REDUCE_KERNEL(i64, long long, bool, false, any)

DEFINE_REDUCE_KERNEL(u8, unsigned char, bool, false, any)
DEFINE_REDUCE_KERNEL(u16, unsigned short, bool, false, any)
DEFINE_REDUCE_KERNEL(u32, unsigned int, bool, false, any)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, bool, false, any)

DEFINE_REDUCE_KERNEL(f32, float, bool, false, any)
DEFINE_REDUCE_KERNEL(f64, double, bool, false, any)

DEFINE_REDUCE_KERNEL(f16, __half, bool, false, any)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, bool, false, any)

// all
#define all1_bool(a, b) (a) && (b)
#define all1_i8(a, b) (a) && (b)
#define all1_i16(a, b) (a) && (b)
#define all1_i32(a, b) (a) && (b)
#define all1_i64(a, b) (a) && (b)
#define all1_f32(a, b) (a) && (b)
#define all1_f64(a, b) (a) && (b)

#define all1_u8(a, b) (a) && (b)
#define all1_u16(a, b) (a) && (b)
#define all1_u32(a, b) (a) && (b)
#define all1_u64(a, b) (a) && (b)

#define all1_f16(a, b) (a) && (b)
#define all1_bf16(a, b) (a) && (b)

#define all1_unary_bool(a) (a)
#define all1_unary_i8(a) ((bool)a)
#define all1_unary_i16(a) ((bool)a)
#define all1_unary_i32(a) ((bool)a)
#define all1_unary_i64(a) ((bool)a)
#define all1_unary_f32(a) ((bool)a)
#define all1_unary_f64(a) ((bool)a)

#define all1_unary_u8(a) ((bool)a)
#define all1_unary_u16(a) ((bool)a)
#define all1_unary_u32(a) ((bool)a)
#define all1_unary_u64(a) ((bool)a)

#define all1_unary_f16(a) ((bool)__half2float(a))
#define all1_unary_bf16(a) ((bool)__bfloat162float(a))

#define all2_bool(a, b) (a) && (b)
#define all2_i8(a, b) (a) && (b)
#define all2_i16(a, b) (a) && (b)
#define all2_i32(a, b) (a) && (b)
#define all2_i64(a, b) (a) && (b)
#define all2_f32(a, b) (a) && (b)
#define all2_f64(a, b) (a) && (b)

#define all2_u8(a, b) (a) && (b)
#define all2_u16(a, b) (a) && (b)
#define all2_u32(a, b) (a) && (b)
#define all2_u64(a, b) (a) && (b)

#define all2_f16(a, b) (a) && (b)
#define all2_bf16(a, b) (a) && (b)

DEFINE_REDUCE_KERNEL(bool, bool, bool, true, all)
DEFINE_REDUCE_KERNEL(i8, char, bool, true, all)
DEFINE_REDUCE_KERNEL(i16, short, bool, true, all)
DEFINE_REDUCE_KERNEL(i32, int, bool, true, all)
DEFINE_REDUCE_KERNEL(i64, long long, bool, true, all)

DEFINE_REDUCE_KERNEL(u8, unsigned char, bool, true, all)
DEFINE_REDUCE_KERNEL(u16, unsigned short, bool, true, all)
DEFINE_REDUCE_KERNEL(u32, unsigned int, bool, true, all)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, bool, true, all)

DEFINE_REDUCE_KERNEL(f32, float, bool, true, all)
DEFINE_REDUCE_KERNEL(f64, double, bool, true, all)

DEFINE_REDUCE_KERNEL(f16, __half, bool, true, all)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, bool, true, all)

// logsumexp
#define logsumexp1_bool(a, b) __float2half_rn(expf((float)(a)) + expf((float)(b)))
#define logsumexp1_i8(a, b) __float2half_rn(expf((float)(a)) + expf((float)(b)))
#define logsumexp1_i16(a, b) __float2half_rn(expf((float)(a)) + expf((float)(b)))
#define logsumexp1_i32(a, b) expf((float)(a)) + expf((float)(b))
#define logsumexp1_i64(a, b) exp((double)(a)) + exp((double)(b))
#define logsumexp1_f32(a, b) expf((a)) + expf((b))
#define logsumexp1_f64(a, b) exp((a)) + exp((b))

#define logsumexp1_u8(a, b) __float2half_rn(expf((float)(a)) + expf((float)(b)))
#define logsumexp1_u16(a, b) __float2half_rn(expf((float)(a)) + expf((float)(b)))
#define logsumexp1_u32(a, b) expf((float)(a)) + expf((float)(b))
#define logsumexp1_u64(a, b) exp((double)(a)) + exp((double)(b))

#define logsumexp1_f16(a, b) __float2half_rn(expf(__half2float(a)) + expf(__half2float(b)))
#define logsumexp1_bf16(a, b) __float2bfloat16_rn(expf(__bfloat162float(a)) + expf(__bfloat162float(b)))

#define logsumexp1_unary_bool(a) __float2half_rn(expf((float)(a)))
#define logsumexp1_unary_i8(a) __float2half_rn(expf((float)(a)))
#define logsumexp1_unary_i16(a) __float2half_rn(expf((float)(a)))
#define logsumexp1_unary_i32(a) expf((float)(a))
#define logsumexp1_unary_i64(a) exp((double)(a))
#define logsumexp1_unary_f32(a) expf(a)
#define logsumexp1_unary_f64(a) exp(a)

#define logsumexp1_unary_u8(a) __float2half_rn(expf((float)(a)))
#define logsumexp1_unary_u16(a) __float2half_rn(expf((float)(a)))
#define logsumexp1_unary_u32(a) expf((float)(a))
#define logsumexp1_unary_u64(a) exp((double)(a))

#define logsumexp1_unary_f16(a) __float2half_rn(expf(__half2float(a)))
#define logsumexp1_unary_bf16(a) __float2bfloat16_rn(expf(__bfloat162float(a)))

#define logsumexp2_bool(a, b) __hadd((a), (b))
#define logsumexp2_i8(a, b) __hadd((a), (b))
#define logsumexp2_i16(a, b) __hadd((a), (b))
#define logsumexp2_i32(a, b) (a) + (b)
#define logsumexp2_i64(a, b) (a) + (b)
#define logsumexp2_f32(a, b) (a) + (b)
#define logsumexp2_f64(a, b) (a) + (b)

#define logsumexp2_u8(a, b) __hadd((a), (b))
#define logsumexp2_u16(a, b) __hadd((a), (b))
#define logsumexp2_u32(a, b) (a) + (b)
#define logsumexp2_u64(a, b) (a) + (b)

#define logsumexp2_f16(a, b) __hadd((a), (b))
#define logsumexp2_bf16(a, b) __hadd((a), (b))

DEFINE_REDUCE_KERNEL(bool, bool, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(i8, char, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(i16, short, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(i32, int, float, 0, logsumexp)
DEFINE_REDUCE_KERNEL(i64, long long, double, 0, logsumexp)

DEFINE_REDUCE_KERNEL(u8, unsigned char, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(u16, unsigned short, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(u32, unsigned int, float, 0, logsumexp)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, double, 0, logsumexp)

DEFINE_REDUCE_KERNEL(f32, float, float, 0.0f, logsumexp)
DEFINE_REDUCE_KERNEL(f64, double, double, 0.0, logsumexp)

DEFINE_REDUCE_KERNEL(f16, __half, __half, 0, logsumexp)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, __hip_bfloat16, 0, logsumexp)