
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <limits.h>
#include <stdio.h>

#define WRAP 32

#define BOOL_MAX 1
#define I8_MAX SCHAR_MAX
#define I16_MAX SHRT_MAX
#define I32_MAX INT_MAX
#define I64_MAX LLONG_MAX
#define F32_MAX INFINITY
#define F64_MAX INFINITY

#define U8_MAX UCHAR_MAX
#define U16_MAX USHRT_MAX
#define U32_MAX UINT_MAX
#define U64_MAX ULONG_MAX
#define F16_MAX __half((unsigned short)31744)
#define BF16_MAX __hip_bfloat16((unsigned short)0x7F80)

#define gt_bool(a, b) ((bool)((unsigned char)a) > ((unsigned char)b))
#define gt_i8(a, b) ((a) > (b))
#define gt_i16(a, b) ((a) > (b))
#define gt_i32(a, b) ((a) > (b))
#define gt_i64(a, b) ((a) > (b))
#define gt_u8(a, b) ((a) > (b))
#define gt_u16(a, b) ((a) > (b))
#define gt_u32(a, b) ((a) > (b))
#define gt_u64(a, b) ((a) > (b))
#define gt_f32(a, b) ((a) > (b))
#define gt_f64(a, b) ((a) > (b))
#define gt_f16(a, b) (__hgt((a), (b)))
#define gt_bf16(a, b) (__hgt((a), (b)))

#define eq_bool(a, b) ((bool)((unsigned char)a) == ((unsigned char)b))
#define eq_i8(a, b) ((a) == (b))
#define eq_i16(a, b) ((a) == (b))
#define eq_i32(a, b) ((a) == (b))
#define eq_i64(a, b) ((a) == (b))
#define eq_u8(a, b) ((a) == (b))
#define eq_u16(a, b) ((a) == (b))
#define eq_u32(a, b) ((a) == (b))
#define eq_u64(a, b) ((a) == (b))
#define eq_f32(a, b) ((a) == (b))
#define eq_f64(a, b) ((a) == (b))
#define eq_f16(a, b) (__heq((a), (b)))
#define eq_bf16(a, b) (__heq((a), (b)))

#define DEFINE_REDUCE_KERNEL(rust_type, type, INIT_VAL)                                                                                                                                                                        \
    __device__ __forceinline__ void warpReduce_##rust_type(volatile type *sdata_##rust_type, volatile long long *sdata_##rust_type_idx, unsigned int tid)                                                                      \
    {                                                                                                                                                                                                                          \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 32]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 32]))      \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 32];                                                                                                                                                      \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 32];                                                                                                                                                              \
        }                                                                                                                                                                                                                      \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 16]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 16]))      \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 16];                                                                                                                                                      \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 16];                                                                                                                                                              \
        }                                                                                                                                                                                                                      \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 8]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 8]))         \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 8];                                                                                                                                                       \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 8];                                                                                                                                                               \
        }                                                                                                                                                                                                                      \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 4]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 4]))         \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 4];                                                                                                                                                       \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 4];                                                                                                                                                               \
        }                                                                                                                                                                                                                      \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 2]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 2]))         \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 2];                                                                                                                                                       \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 2];                                                                                                                                                               \
        }                                                                                                                                                                                                                      \
        if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + 1]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + 1]))         \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + 1];                                                                                                                                                       \
            sdata_##rust_type[tid] = sdata_##rust_type[tid + 1];                                                                                                                                                               \
        }                                                                                                                                                                                                                      \
    }                                                                                                                                                                                                                          \
    extern "C" __global__ void contiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, size_t size)                                                                                                             \
    {                                                                                                                                                                                                                          \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                                                            \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                                                        \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                        \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                            \
        sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                                     \
        if (i + blockDim.x < size)                                                                                                                                                                                             \
        {                                                                                                                                                                                                                      \
            if (gt_##rust_type(in[i], in[i + blockDim.x]))                                                                                                                                                                     \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                                                   \
                sdata_##rust_type_idx[tid] = (long long)(i + blockDim.x);                                                                                                                                                      \
            }                                                                                                                                                                                                                  \
            else                                                                                                                                                                                                               \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                                                \
                sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                   \
            }                                                                                                                                                                                                                  \
        }                                                                                                                                                                                                                      \
        else if (i < size)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                                                    \
            sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                       \
        }                                                                                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                                                       \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                               \
        {                                                                                                                                                                                                                      \
            if (tid < s)                                                                                                                                                                                                       \
            {                                                                                                                                                                                                                  \
                if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + s])) \
                {                                                                                                                                                                                                              \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                                                       \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                                               \
                }                                                                                                                                                                                                              \
            }                                                                                                                                                                                                                  \
            __syncthreads();                                                                                                                                                                                                   \
        }                                                                                                                                                                                                                      \
        if (tid < WRAP)                                                                                                                                                                                                        \
        {                                                                                                                                                                                                                      \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                                             \
        }                                                                                                                                                                                                                      \
        if (tid == 0)                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                      \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                                                            \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                                                    \
        }                                                                                                                                                                                                                      \
    }                                                                                                                                                                                                                          \
    extern "C" __global__ void uncontiguous_reduce_##rust_type(type *out, long long *out_idx, type *in, long long *shape, long long *strides, size_t ndim, size_t size)                                                        \
    {                                                                                                                                                                                                                          \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                                                            \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                                                        \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                        \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                            \
        sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                                     \
        if (i + blockDim.x < size)                                                                                                                                                                                             \
        {                                                                                                                                                                                                                      \
            long long a_offset = 0;                                                                                                                                                                                            \
            long long a_amount = i;                                                                                                                                                                                            \
            long long b_offset = 0;                                                                                                                                                                                            \
            long long b_amount = i + blockDim.x;                                                                                                                                                                               \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                                \
            {                                                                                                                                                                                                                  \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                                \
                a_amount /= shape[j];                                                                                                                                                                                          \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                                                \
                b_amount /= shape[j];                                                                                                                                                                                          \
            }                                                                                                                                                                                                                  \
            if (gt_##rust_type(in[a_offset], in[b_offset]))                                                                                                                                                                    \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                                         \
                sdata_##rust_type_idx[tid] = (long long)(i + blockDim.x);                                                                                                                                                      \
            }                                                                                                                                                                                                                  \
            else                                                                                                                                                                                                               \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                                         \
                sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                   \
            }                                                                                                                                                                                                                  \
        }                                                                                                                                                                                                                      \
        else if (i < size)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                      \
            long long a_offset = 0;                                                                                                                                                                                            \
            long long a_amount = i;                                                                                                                                                                                            \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                                \
            {                                                                                                                                                                                                                  \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                                \
                a_amount /= shape[j];                                                                                                                                                                                          \
            }                                                                                                                                                                                                                  \
            sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                                             \
            sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                       \
        }                                                                                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                                                       \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                               \
        {                                                                                                                                                                                                                      \
            if (tid < s)                                                                                                                                                                                                       \
            {                                                                                                                                                                                                                  \
                if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + s])) \
                {                                                                                                                                                                                                              \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                                                       \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                                               \
                }                                                                                                                                                                                                              \
            }                                                                                                                                                                                                                  \
            __syncthreads();                                                                                                                                                                                                   \
        }                                                                                                                                                                                                                      \
        if (tid < WRAP)                                                                                                                                                                                                        \
        {                                                                                                                                                                                                                      \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                                             \
        }                                                                                                                                                                                                                      \
        if (tid == 0)                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                      \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                                                            \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                                                    \
        }                                                                                                                                                                                                                      \
    }                                                                                                                                                                                                                          \
    extern "C" __global__ void contiguous_reduce2_##rust_type(type *out, long long *out_idx, type *in, size_t size)                                                                                                            \
    {                                                                                                                                                                                                                          \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                                                            \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                                                        \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                        \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                            \
        sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                           \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                                     \
        if (i + blockDim.x < size)                                                                                                                                                                                             \
        {                                                                                                                                                                                                                      \
            if (gt_##rust_type(in[i], in[i + blockDim.x]) || (eq_##rust_type(in[i], in[i + blockDim.x]) && out_idx[i] > out_idx[i + blockDim.x]))                                                                              \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[i + blockDim.x];                                                                                                                                                                   \
                sdata_##rust_type_idx[tid] = out_idx[i + blockDim.x];                                                                                                                                                          \
            }                                                                                                                                                                                                                  \
            else                                                                                                                                                                                                               \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[i];                                                                                                                                                                                \
                sdata_##rust_type_idx[tid] = out_idx[i];                                                                                                                                                                       \
            }                                                                                                                                                                                                                  \
        }                                                                                                                                                                                                                      \
        else if (i < size)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type[tid] = in[i];                                                                                                                                                                                    \
            sdata_##rust_type_idx[tid] = out_idx[i];                                                                                                                                                                           \
        }                                                                                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                                                       \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                               \
        {                                                                                                                                                                                                                      \
            if (tid < s)                                                                                                                                                                                                       \
            {                                                                                                                                                                                                                  \
                if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + s])) \
                {                                                                                                                                                                                                              \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                                                       \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                                               \
                }                                                                                                                                                                                                              \
            }                                                                                                                                                                                                                  \
            __syncthreads();                                                                                                                                                                                                   \
        }                                                                                                                                                                                                                      \
        if (tid < WRAP)                                                                                                                                                                                                        \
        {                                                                                                                                                                                                                      \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                                             \
        }                                                                                                                                                                                                                      \
        if (tid == 0)                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                      \
            out[blockIdx.x] = sdata_##rust_type[0];                                                                                                                                                                            \
            out_idx[blockIdx.x] = sdata_##rust_type_idx[0];                                                                                                                                                                    \
        }                                                                                                                                                                                                                      \
    }                                                                                                                                                                                                                          \
    extern "C" __global__ void nkd_##rust_type(type *out, type *in, long long *out_idx, long long *shape, long long *strides, size_t ndim, size_t start_row_idx, size_t cols, size_t rows, size_t num_blocks_per_row)          \
    {                                                                                                                                                                                                                          \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                                                            \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                                                        \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                        \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                            \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                                     \
        sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                           \
        if (blockIdx.y + start_row_idx >= rows)                                                                                                                                                                                \
        {                                                                                                                                                                                                                      \
            return;                                                                                                                                                                                                            \
        }                                                                                                                                                                                                                      \
        if (i + blockDim.x < cols)                                                                                                                                                                                             \
        {                                                                                                                                                                                                                      \
            long long a_offset = 0;                                                                                                                                                                                            \
            long long a_amount = i + (blockIdx.y + start_row_idx) * cols;                                                                                                                                                      \
            long long b_offset = 0;                                                                                                                                                                                            \
            long long b_amount = i + blockDim.x + (blockIdx.y + start_row_idx) * cols;                                                                                                                                         \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                                \
            {                                                                                                                                                                                                                  \
                a_offset += (a_amount % shape[j]) * strides[j];                                                                                                                                                                \
                a_amount /= shape[j];                                                                                                                                                                                          \
                b_offset += (b_amount % shape[j]) * strides[j];                                                                                                                                                                \
                b_amount /= shape[j];                                                                                                                                                                                          \
            }                                                                                                                                                                                                                  \
            if (gt_##rust_type(in[a_offset], in[b_offset]))                                                                                                                                                                    \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[b_offset];                                                                                                                                                                         \
                sdata_##rust_type_idx[tid] = (long long)(i + blockDim.x);                                                                                                                                                      \
            }                                                                                                                                                                                                                  \
            else                                                                                                                                                                                                               \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[a_offset];                                                                                                                                                                         \
                sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                   \
            }                                                                                                                                                                                                                  \
        }                                                                                                                                                                                                                      \
        else if (i < cols)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                      \
            long long a_amount = i + (blockIdx.y + start_row_idx) * cols;                                                                                                                                                      \
            for (int j = ndim - 1; j >= 0; j--)                                                                                                                                                                                \
            {                                                                                                                                                                                                                  \
                in += (a_amount % shape[j]) * strides[j];                                                                                                                                                                      \
                a_amount /= shape[j];                                                                                                                                                                                          \
            }                                                                                                                                                                                                                  \
            sdata_##rust_type[tid] = *in;                                                                                                                                                                                      \
            sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                       \
        }                                                                                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                                                       \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                               \
        {                                                                                                                                                                                                                      \
            if (tid < s)                                                                                                                                                                                                       \
            {                                                                                                                                                                                                                  \
                if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + s])) \
                {                                                                                                                                                                                                              \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                                                       \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                                               \
                }                                                                                                                                                                                                              \
            }                                                                                                                                                                                                                  \
            __syncthreads();                                                                                                                                                                                                   \
        }                                                                                                                                                                                                                      \
        if (tid < WRAP)                                                                                                                                                                                                        \
        {                                                                                                                                                                                                                      \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                                             \
        }                                                                                                                                                                                                                      \
        if (tid == 0)                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                      \
            out[blockIdx.x + (blockIdx.y + start_row_idx) * num_blocks_per_row] = sdata_##rust_type[0];                                                                                                                        \
            out_idx[blockIdx.x + (blockIdx.y + start_row_idx) * num_blocks_per_row] = sdata_##rust_type_idx[0];                                                                                                                \
        }                                                                                                                                                                                                                      \
    }                                                                                                                                                                                                                          \
    extern "C" __global__ void nkd2_##rust_type(type *out, type *in, long long *out_idx, size_t start_row_idx, size_t cols, size_t rows, size_t num_blocks_per_row, size_t original_cols)                                      \
    {                                                                                                                                                                                                                          \
        extern __shared__ type sdata_##rust_type[];                                                                                                                                                                            \
        long long *sdata_##rust_type_idx = (long long *)&sdata_##rust_type[blockDim.x];                                                                                                                                        \
        unsigned int tid = threadIdx.x;                                                                                                                                                                                        \
        unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;                                                                                                                                                            \
        sdata_##rust_type[tid] = INIT_VAL;                                                                                                                                                                                     \
        sdata_##rust_type_idx[tid] = (long long)(i);                                                                                                                                                                           \
        if (blockIdx.y + start_row_idx >= rows)                                                                                                                                                                                \
        {                                                                                                                                                                                                                      \
            return;                                                                                                                                                                                                            \
        }                                                                                                                                                                                                                      \
        unsigned int a_idx = i + (blockIdx.y + start_row_idx) * original_cols;                                                                                                                                                 \
        unsigned int b_idx = i + blockDim.x + (blockIdx.y + start_row_idx) * original_cols;                                                                                                                                    \
        if (i + blockDim.x < cols)                                                                                                                                                                                             \
        {                                                                                                                                                                                                                      \
            if (gt_##rust_type(in[a_idx], in[b_idx]) || (eq_##rust_type(in[a_idx], in[b_idx]) && out_idx[a_idx] > out_idx[b_idx]))                                                                                             \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[b_idx];                                                                                                                                                                            \
                sdata_##rust_type_idx[tid] = out_idx[b_idx];                                                                                                                                                                   \
            }                                                                                                                                                                                                                  \
            else                                                                                                                                                                                                               \
            {                                                                                                                                                                                                                  \
                sdata_##rust_type[tid] = in[a_idx];                                                                                                                                                                            \
                sdata_##rust_type_idx[tid] = out_idx[a_idx];                                                                                                                                                                   \
            }                                                                                                                                                                                                                  \
        }                                                                                                                                                                                                                      \
        else if (i < cols)                                                                                                                                                                                                     \
        {                                                                                                                                                                                                                      \
            sdata_##rust_type[tid] = in[a_idx];                                                                                                                                                                                \
            sdata_##rust_type_idx[tid] = out_idx[a_idx];                                                                                                                                                                       \
        }                                                                                                                                                                                                                      \
        __syncthreads();                                                                                                                                                                                                       \
        for (unsigned int s = blockDim.x / 2; s > WRAP; s >>= 1)                                                                                                                                                               \
        {                                                                                                                                                                                                                      \
            if (tid < s)                                                                                                                                                                                                       \
            {                                                                                                                                                                                                                  \
                if (gt_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) || (eq_##rust_type(sdata_##rust_type[tid], sdata_##rust_type[tid + s]) && sdata_##rust_type_idx[tid] > sdata_##rust_type_idx[tid + s])) \
                {                                                                                                                                                                                                              \
                    sdata_##rust_type[tid] = sdata_##rust_type[tid + s];                                                                                                                                                       \
                    sdata_##rust_type_idx[tid] = sdata_##rust_type_idx[tid + s];                                                                                                                                               \
                }                                                                                                                                                                                                              \
            }                                                                                                                                                                                                                  \
            __syncthreads();                                                                                                                                                                                                   \
        }                                                                                                                                                                                                                      \
        if (tid < WRAP)                                                                                                                                                                                                        \
        {                                                                                                                                                                                                                      \
            warpReduce_##rust_type(sdata_##rust_type, sdata_##rust_type_idx, tid);                                                                                                                                             \
        }                                                                                                                                                                                                                      \
        if (tid == 0)                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                      \
            out[blockIdx.x + (blockIdx.y + start_row_idx) * original_cols] = sdata_##rust_type[0];                                                                                                                             \
            out_idx[blockIdx.x + (blockIdx.y + start_row_idx) * original_cols] = sdata_##rust_type_idx[0];                                                                                                                     \
        }                                                                                                                                                                                                                      \
    }

DEFINE_REDUCE_KERNEL(bool, bool, BOOL_MAX)
DEFINE_REDUCE_KERNEL(i8, char, I8_MAX)
DEFINE_REDUCE_KERNEL(i16, short, I16_MAX)
DEFINE_REDUCE_KERNEL(i32, int, I32_MAX)
DEFINE_REDUCE_KERNEL(i64, long long, I64_MAX)
DEFINE_REDUCE_KERNEL(u8, unsigned char, U8_MAX)
DEFINE_REDUCE_KERNEL(u16, unsigned short, U16_MAX)
DEFINE_REDUCE_KERNEL(u32, unsigned int, U32_MAX)
DEFINE_REDUCE_KERNEL(u64, unsigned long long, U64_MAX)
DEFINE_REDUCE_KERNEL(f32, float, F32_MAX)
DEFINE_REDUCE_KERNEL(f64, double, F64_MAX)
DEFINE_REDUCE_KERNEL(f16, __half, F16_MAX)
DEFINE_REDUCE_KERNEL(bf16, __hip_bfloat16, BF16_MAX)
