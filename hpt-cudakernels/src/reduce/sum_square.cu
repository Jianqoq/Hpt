#include "hip/hip_runtime.h"
#include "declare_macros.cuh"

template <typename T, typename R, unsigned int WarpSize = 32>
class SumSquare : public ReduceOp<T, R, WarpSize>
{
public:
    __device__ __forceinline__ static T combine(T a, T b)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hadd(a, b);
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a || b;
        }
        else
        {
            return a + b;
        }
    }

    __device__ __forceinline__ static T identity()
    {
        if constexpr (std::is_same_v<T, float>)
        {
            return 0.0f;
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return 0.0;
        }
        else if constexpr (std::is_same_v<T, __half>)
        {
            return __float2half(0.0f);
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16>)
        {
            return __float2bfloat16(0.0f);
        }
        else
        {
            return T(0);
        }
    }

    __device__ __forceinline__ static R warp_reduce(R a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            a += __shfl_down_sync(0xffffffff, a, mask);
        }
        return a;
    }

    __device__ __forceinline__ static R pre_op(T a)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hmul(a, a);
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a && a;
        }
        else
        {
            return a * a;
        }
    }
};

DECLARE_KERNEL(bool, bool, bool, sumsquare, SumSquare)
DECLARE_KERNEL(uint8_t, uint8_t, u8, sumsquare, SumSquare)
DECLARE_KERNEL(uint16_t, uint16_t, u16, sumsquare, SumSquare)
DECLARE_KERNEL(uint32_t, uint32_t, u32, sumsquare, SumSquare)
DECLARE_KERNEL(uint64_t, uint64_t, u64, sumsquare, SumSquare)
DECLARE_KERNEL(int8_t, int8_t, i8, sumsquare, SumSquare)
DECLARE_KERNEL(int16_t, int16_t, i16, sumsquare, SumSquare)
DECLARE_KERNEL(int32_t, int32_t, i32, sumsquare, SumSquare)
DECLARE_KERNEL(int64_t, int64_t, i64, sumsquare, SumSquare)
DECLARE_KERNEL(float, float, f32, sumsquare, SumSquare)
DECLARE_KERNEL(double, double, f64, sumsquare, SumSquare)
DECLARE_KERNEL(__half, __half, f16, sumsquare, SumSquare)
DECLARE_KERNEL(__hip_bfloat16, __hip_bfloat16, bf16, sumsquare, SumSquare)
