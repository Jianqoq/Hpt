#include "hip/hip_runtime.h"
#include "declare_macros.cuh"
#include "../utils/type_cast.cuh"
#include "../utils/check_type.cuh"

template <typename T, typename R, unsigned int WarpSize = 32>
class ReduceL3 : public ReduceOp<T, R, WarpSize>
{
public:
    __device__ __forceinline__ static T combine(T a, T b)
    {
        return a + b;
    }

    __device__ __forceinline__ static T identity()
    {
        if constexpr (std::is_same_v<T, float>)
        {
            return 0.0f;
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return 0.0;
        }
        else if constexpr (std::is_same_v<T, __half>)
        {
            return __float2half(0.0f);
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16>)
        {
            return __float2bfloat16(0.0f);
        }
        else
        {
            return T(0);
        }
    }

    __device__ __forceinline__ static R warp_reduce(R a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            a += __shfl_down_sync(0xffffffff, a, mask);
        }
        return a;
    }

    __device__ __forceinline__ static R pre_op(T a)
    {
        R res = cast<T, R>(a);
        CHECK_FLOAT_TYPE(R);
        if constexpr (std::is_same_v<R, half> || std::is_same_v<R, __hip_bfloat16>)
        {
            R abs = __habs(res);
            return abs * abs * abs;
        }
        else if constexpr (std::is_same_v<R, float>)
        {
            float abs = fabsf(res);
            return abs * abs * abs;
        }
        else if constexpr (std::is_same_v<R, double>)
        {
            double abs = fabs(res);
            return abs * abs * abs;
        }
        else
        {
            return res;
        }
    }

    __device__ __forceinline__ static R post_op(R a, size_t count)
    {
        if constexpr (std::is_same_v<R, half>)
        {
            return __float2half(cbrtf(__half2float(a)));
        }
        else if constexpr (std::is_same_v<R, __hip_bfloat16>)
        {
            return __float2bfloat16(cbrtf(__bfloat162float(a)));
        }
        else if constexpr (std::is_same_v<R, float>)
        {
            return cbrtf(a);
        }
        else if constexpr (std::is_same_v<R, double>)
        {
            return cbrt(a);
        }
        else
        {
            return a;
        }
    }
};

DECLARE_KERNEL(half, bool, bool, reducel3, ReduceL3)
DECLARE_KERNEL(half, uint8_t, u8, reducel3, ReduceL3)
DECLARE_KERNEL(half, uint16_t, u16, reducel3, ReduceL3)
DECLARE_KERNEL(float, uint32_t, u32, reducel3, ReduceL3)
DECLARE_KERNEL(double, uint64_t, u64, reducel3, ReduceL3)
DECLARE_KERNEL(half, int8_t, i8, reducel3, ReduceL3)
DECLARE_KERNEL(half, int16_t, i16, reducel3, ReduceL3)
DECLARE_KERNEL(float, int32_t, i32, reducel3, ReduceL3)
DECLARE_KERNEL(double, int64_t, i64, reducel3, ReduceL3)
DECLARE_KERNEL(float, float, f32, reducel3, ReduceL3)
DECLARE_KERNEL(double, double, f64, reducel3, ReduceL3)
DECLARE_KERNEL(half, half, f16, reducel3, ReduceL3)
DECLARE_KERNEL(__hip_bfloat16, __hip_bfloat16, bf16, reducel3, ReduceL3)
