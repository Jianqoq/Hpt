#include "hip/hip_runtime.h"
#include "declare_macros.cuh"
#include "../utils/type_utils.cuh"
#include "reduce_classes.cuh"

DECLARE_KERNEL(bool, bool, max, Max)
DECLARE_KERNEL(u8, u8, max, Max)
DECLARE_KERNEL(u16, u16, max, Max)
DECLARE_KERNEL(u32, u32, max, Max)
DECLARE_KERNEL(u64, u64, max, Max)
DECLARE_KERNEL(i8, i8, max, Max)
DECLARE_KERNEL(i16, i16, max, Max)
DECLARE_KERNEL(i32, i32, max, Max)
DECLARE_KERNEL(i64, i64, max, Max)
DECLARE_KERNEL(f32, f32, max, Max)
DECLARE_KERNEL(f64, f64, max, Max)
DECLARE_KERNEL(f16, f16, max, Max)
DECLARE_KERNEL(bf16, bf16, max, Max)
