#include "hip/hip_runtime.h"
#include "declare_macros.cuh"

template <typename T, typename R, unsigned int WarpSize = 32>
class Prod : public ReduceOp<T, R, WarpSize>
{
public:
    __device__ __forceinline__ static T combine(T a, T b)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hmul(a, b);
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a && b;
        }
        else
        {
            return a * b;
        }
    }

    __device__ __forceinline__ static T identity()
    {
        if constexpr (std::is_same_v<T, float>)
        {
            return 1.0f;
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return 1.0;
        }
        else if constexpr (std::is_same_v<T, __half>)
        {
            return __float2half(1.0f);
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16>)
        {
            return __float2bfloat16(1.0f);
        }
        else
        {
            return T(1);
        }
    }

    __device__ __forceinline__ static R warp_reduce(R a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            a *= __shfl_down_sync(0xffffffff, a, mask);
        }
        return a;
    }
};

DECLARE_KERNEL(bool, bool, bool, prod, Prod)
DECLARE_KERNEL(uint8_t, uint8_t, u8, prod, Prod)
DECLARE_KERNEL(uint16_t, uint16_t, u16, prod, Prod)
DECLARE_KERNEL(uint32_t, uint32_t, u32, prod, Prod)
DECLARE_KERNEL(uint64_t, uint64_t, u64, prod, Prod)
DECLARE_KERNEL(int8_t, int8_t, i8, prod, Prod)
DECLARE_KERNEL(int16_t, int16_t, i16, prod, Prod)
DECLARE_KERNEL(int32_t, int32_t, i32, prod, Prod)
DECLARE_KERNEL(int64_t, int64_t, i64, prod, Prod)
DECLARE_KERNEL(float, float, f32, prod, Prod)
DECLARE_KERNEL(double, double, f64, prod, Prod)
DECLARE_KERNEL(__half, __half, f16, prod, Prod)
DECLARE_KERNEL(__hip_bfloat16, __hip_bfloat16, bf16, prod, Prod)
