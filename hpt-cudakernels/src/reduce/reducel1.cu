#include "hip/hip_runtime.h"
#include "declare_macros.cuh"
#include "../utils/type_cast.cuh"

template <typename T, typename R, unsigned int WarpSize = 32>
class ReduceL1 : public ReduceOp<T, R, WarpSize>
{
public:
    __device__ __forceinline__ static T combine(T a, T b)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hadd(a, b);
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a || b;
        }
        else
        {
            return a + b;
        }
    }

    __device__ __forceinline__ static T identity()
    {
        if constexpr (std::is_same_v<T, float>)
        {
            return 0.0f;
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return 0.0;
        }
        else if constexpr (std::is_same_v<T, __half>)
        {
            return __float2half(0.0f);
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16>)
        {
            return __float2bfloat16(0.0f);
        }
        else
        {
            return T(0);
        }
    }

    __device__ __forceinline__ static R warp_reduce(R a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            a += __shfl_down_sync(0xffffffff, a, mask);
        }
        return a;
    }

    __device__ __forceinline__ static R pre_op(T a)
    {
        R res = cast<T, R>(a);
        if constexpr (std::is_same_v<R, half> || std::is_same_v<R, __hip_bfloat16>)
        {
            return __habs(res);
        }
        else if constexpr (std::is_same_v<R, float>)
        {
            return fabsf(res);
        }
        else if constexpr (std::is_same_v<R, double>)
        {
            return fabs(res);
        }
        else if constexpr (std::is_signed_v<R>)
        {
            return res < 0 ? -res : res;
        }
        else
        {
            return res;
        }
    }
};

DECLARE_KERNEL(bool, bool, bool, reducel1, ReduceL1)
DECLARE_KERNEL(uint8_t, uint8_t, u8, reducel1, ReduceL1)
DECLARE_KERNEL(uint16_t, uint16_t, u16, reducel1, ReduceL1)
DECLARE_KERNEL(uint32_t, uint32_t, u32, reducel1, ReduceL1)
DECLARE_KERNEL(uint64_t, uint64_t, u64, reducel1, ReduceL1)
DECLARE_KERNEL(int8_t, int8_t, i8, reducel1, ReduceL1)
DECLARE_KERNEL(int16_t, int16_t, i16, reducel1, ReduceL1)
DECLARE_KERNEL(int32_t, int32_t, i32, reducel1, ReduceL1)
DECLARE_KERNEL(int64_t, int64_t, i64, reducel1, ReduceL1)
DECLARE_KERNEL(float, float, f32, reducel1, ReduceL1)
DECLARE_KERNEL(double, double, f64, reducel1, ReduceL1)
DECLARE_KERNEL(__half, __half, f16, reducel1, ReduceL1)
DECLARE_KERNEL(__hip_bfloat16, __hip_bfloat16, bf16, reducel1, ReduceL1)
