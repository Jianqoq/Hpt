#include "hip/hip_runtime.h"
#include "arg_template.cuh"
#include "../utils/type_utils.cuh"
#include "declare_macros.cuh"
#include <stdio.h>
template <typename T>
struct ArgMinResult
{
    T val;
    int64_t idx;

    __device__ __forceinline__ ArgMinResult(T val, int64_t idx) : val(val), idx(idx) {}

    __device__ __forceinline__ static ArgMinResult identity()
    {
        return ArgMinResult();
    }

    __device__ __forceinline__ ArgMinResult() : val(TypeUtils<T>::limit_max()), idx(0) {}
};

template <typename T, unsigned int WarpSize = 32>
class ArgMin
{
public:
    __device__ __forceinline__ static ArgMinResult<T> combine(ArgMinResult<T> a, ArgMinResult<T> b)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hlt(a.val, b.val) ? a : (__hgt(a.val, b.val) ? b : (a.idx < b.idx ? a : b));
        }
        else if constexpr (std::is_same_v<T, float>)
        {
            return a.val < b.val ? a : (a.val > b.val ? b : (a.idx < b.idx ? a : b));
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return a.val < b.val ? a : (a.val > b.val ? b : (a.idx < b.idx ? a : b));
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a.val < b.val ? a : (a.val > b.val ? b : (a.idx < b.idx ? a : b));
        }
        else
        {
            return a.val < b.val ? a : (a.val > b.val ? b : (a.idx < b.idx ? a : b));
        }
    }

    __device__ __forceinline__ static ArgMinResult<T> identity()
    {
        return ArgMinResult<T>::identity();
    }

    __device__ __forceinline__ static ArgMinResult<T> warp_reduce(ArgMinResult<T> a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            T value = __shfl_down_sync(0xffffffff, a.val, mask);
            int64_t index = __shfl_down_sync(0xffffffff, a.idx, mask);
            ArgMinResult<T> other{value, index};
            a = ArgMin<T, WarpSize>::combine(a, other);
        }
        return a;
    }
};

DECLARE_ARG_KERNEL(bool, bool, argmin, ArgMin);
DECLARE_ARG_KERNEL(int8_t, i8, argmin, ArgMin);
DECLARE_ARG_KERNEL(int16_t, i16, argmin, ArgMin);
DECLARE_ARG_KERNEL(int32_t, i32, argmin, ArgMin);
DECLARE_ARG_KERNEL(int64_t, i64, argmin, ArgMin);
DECLARE_ARG_KERNEL(uint8_t, u8, argmin, ArgMin);
DECLARE_ARG_KERNEL(uint16_t, u16, argmin, ArgMin);
DECLARE_ARG_KERNEL(uint32_t, u32, argmin, ArgMin);
DECLARE_ARG_KERNEL(uint64_t, u64, argmin, ArgMin);
DECLARE_ARG_KERNEL(__half, f16, argmin, ArgMin);
DECLARE_ARG_KERNEL(float, f32, argmin, ArgMin);
DECLARE_ARG_KERNEL(double, f64, argmin, ArgMin);
DECLARE_ARG_KERNEL(__hip_bfloat16, bf16, argmin, ArgMin);
