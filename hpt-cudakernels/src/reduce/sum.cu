#include "hip/hip_runtime.h"
#include "declare_macros.cuh"

template <typename T, typename R, unsigned int WarpSize = 32>
class Sum : public ReduceOp<T, R, WarpSize>
{
public:
    __device__ __forceinline__ static T combine(T a, T b)
    {
        if constexpr (std::is_same_v<T, __half> || std::is_same_v<T, __hip_bfloat16>)
        {
            return __hadd(a, b);
        }
        else if constexpr (std::is_same_v<T, bool>)
        {
            return a || b;
        }
        else
        {
            return a + b;
        }
    }

    __device__ __forceinline__ static T identity()
    {
        if constexpr (std::is_same_v<T, float>)
        {
            return 0.0f;
        }
        else if constexpr (std::is_same_v<T, double>)
        {
            return 0.0;
        }
        else if constexpr (std::is_same_v<T, __half>)
        {
            return __float2half(0.0f);
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16>)
        {
            return __float2bfloat16(0.0f);
        }
        else
        {
            return T(0);
        }
    }

    __device__ __forceinline__ static R warp_reduce(R a)
    {
#pragma unroll
        for (int mask = WarpSize >> 1; mask >= 1; mask >>= 1)
        {
            a += __shfl_down_sync(0xffffffff, a, mask);
        }
        return a;
    }
};

DECLARE_KERNEL(bool, bool, bool, sum, Sum)
DECLARE_KERNEL(uint8_t, uint8_t, u8, sum, Sum)
DECLARE_KERNEL(uint16_t, uint16_t, u16, sum, Sum)
DECLARE_KERNEL(uint32_t, uint32_t, u32, sum, Sum)
DECLARE_KERNEL(uint64_t, uint64_t, u64, sum, Sum)
DECLARE_KERNEL(int8_t, int8_t, i8, sum, Sum)
DECLARE_KERNEL(int16_t, int16_t, i16, sum, Sum)
DECLARE_KERNEL(int32_t, int32_t, i32, sum, Sum)
DECLARE_KERNEL(int64_t, int64_t, i64, sum, Sum)
DECLARE_KERNEL(float, float, f32, sum, Sum)
DECLARE_KERNEL(double, double, f64, sum, Sum)
DECLARE_KERNEL(__half, __half, f16, sum, Sum)
DECLARE_KERNEL(__hip_bfloat16, __hip_bfloat16, bf16, sum, Sum)
