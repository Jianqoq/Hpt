#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#define DEFINE_LINSPACE_KERNEL(func_name, vec_type, type, vec_size)                  \
    extern "C" __global__ void func_name(type *out, type start, type step, size_t n) \
    {                                                                                \
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;                          \
        size_t stride = blockDim.x * gridDim.x;                                      \
                                                                                     \
        for (size_t i = idx; i < n / vec_size; i += stride)                          \
        {                                                                            \
            ##vec_type##vec_size value;                                              \
            value.x = start + step * (i * vec_size + 0);                             \
            value.y = start + step * (i * vec_size + 1);                             \
            value.z = start + step * (i * vec_size + 2);                             \
            value.w = start + step * (i * vec_size + 3);                             \
            ((##vec_type##vec_size *)out)[i] = value;                                \
        }                                                                            \
                                                                                     \
        if (idx == 0)                                                                \
        {                                                                            \
            for (size_t i = (n / vec_size) * vec_size; i < n; i++)                   \
            {                                                                        \
                out[i] = start + step * i;                                           \
            }                                                                        \
        }                                                                            \
    }
extern "C" __global__ void linspace_f16_vec2(half *out, float start, float step, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < n / 2; i += stride)
    {
        half2 value;
        value.x = start + step * (i * 2 + 0);
        value.y = start + step * (i * 2 + 1);
        ((half2 *)out)[i] = value;
    }
    if (idx == 0)
    {
        for (size_t i = (n / 2) * 2; i < n; i++)
        {
            out[i] = start + step * i;
        }
    }
};
DEFINE_LINSPACE_KERNEL(linspace_f32_vec4, float, float, 4);
DEFINE_LINSPACE_KERNEL(linspace_f64_vec4, double, double, 4);

DEFINE_LINSPACE_KERNEL(linspace_i8_vec4, char, char, 4);
DEFINE_LINSPACE_KERNEL(linspace_i16_vec4, short, short, 4);
DEFINE_LINSPACE_KERNEL(linspace_i32_vec4, int, int, 4);
DEFINE_LINSPACE_KERNEL(linspace_i64_vec4, longlong, long long, 4);

DEFINE_LINSPACE_KERNEL(linspace_u8_vec4, uchar, unsigned char, 4);
DEFINE_LINSPACE_KERNEL(linspace_u16_vec4, ushort, unsigned short, 4);
DEFINE_LINSPACE_KERNEL(linspace_u32_vec4, uint, unsigned int, 4);
DEFINE_LINSPACE_KERNEL(linspace_u64_vec4, ulonglong, unsigned long long, 4);