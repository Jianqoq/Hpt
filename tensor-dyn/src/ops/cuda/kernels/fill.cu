
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define MAKE_VEC4(vec_type, vec_size, value) make_##vec_type##vec_size(value, value, value, value)
#define MAKE_VEC3(vec_type, vec_size, value) make_##vec_type##vec_size(value, value, value)
#define MAKE_VEC2(vec_type, vec_size, value) make_##vec_type##vec_size(value, value)

#define DEFINE_FILL_KERNEL(func_name, vec_type, type, vec_size)                       \
    extern "C" __global__ void func_name(type *out, type value, size_t N)                        \
    {                                                                                 \
        vec_type##vec_size *out_vec = (vec_type##vec_size *)out;                      \
        vec_type##vec_size value_vec = MAKE_VEC##vec_size(vec_type, vec_size, value); \
                                                                                      \
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;                           \
        size_t stride = blockDim.x * gridDim.x;                                       \
        size_t N_vec = N / vec_size;                                                  \
                                                                                      \
        for (size_t i = idx; i < N_vec; i += stride)                                  \
        {                                                                             \
            out_vec[i] = value_vec;                                                   \
        }                                                                             \
                                                                                      \
        if (idx == 0)                                                                 \
        {                                                                             \
            for (size_t i = N_vec * vec_size; i < N; i++)                             \
            {                                                                         \
                out[i] = value;                                                       \
            }                                                                         \
        }                                                                             \
    }

extern "C" __global__ void fill_f16_vec2(half *out, half value, size_t N)
{
    __half2 *out_vec = (__half2 *)out;
    __half2 value_vec = make_half2(value, value);
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t N_vec = N / 2;
    for (size_t i = idx; i < N_vec; i += stride)
    {
        out_vec[i] = value_vec;
    }
    if (idx == 0)
    {
        for (size_t i = N_vec * 2; i < N; i++)
        {
            out[i] = value;
        }
    }
};
DEFINE_FILL_KERNEL(fill_f32_vec4, float, float, 4);
DEFINE_FILL_KERNEL(fill_f32_vec3, float, float, 3);
DEFINE_FILL_KERNEL(fill_f32_vec2, float, float, 2);
DEFINE_FILL_KERNEL(fill_f64_vec4, double, double, 4);
DEFINE_FILL_KERNEL(fill_f64_vec3, double, double, 3);
DEFINE_FILL_KERNEL(fill_f64_vec2, double, double, 2);

DEFINE_FILL_KERNEL(fill_i8_vec4, char, char, 4);
DEFINE_FILL_KERNEL(fill_i8_vec3, char, char, 3);
DEFINE_FILL_KERNEL(fill_i8_vec2, char, char, 2);
DEFINE_FILL_KERNEL(fill_i16_vec4, short, short, 4);
DEFINE_FILL_KERNEL(fill_i16_vec3, short, short, 3);
DEFINE_FILL_KERNEL(fill_i16_vec2, short, short, 2);
DEFINE_FILL_KERNEL(fill_i32_vec4, int, int, 4);
DEFINE_FILL_KERNEL(fill_i32_vec3, int, int, 3);
DEFINE_FILL_KERNEL(fill_i32_vec2, int, int, 2);
DEFINE_FILL_KERNEL(fill_i64_vec4, longlong, long long, 4);
DEFINE_FILL_KERNEL(fill_i64_vec3, longlong, long long, 3);
DEFINE_FILL_KERNEL(fill_i64_vec2, longlong, long long, 2);

DEFINE_FILL_KERNEL(fill_u8_vec4, uchar, unsigned char, 4);
DEFINE_FILL_KERNEL(fill_u8_vec3, uchar, unsigned char, 3);
DEFINE_FILL_KERNEL(fill_u8_vec2, uchar, unsigned char, 2);
DEFINE_FILL_KERNEL(fill_u16_vec4, ushort, unsigned short, 4);
DEFINE_FILL_KERNEL(fill_u16_vec3, ushort, unsigned short, 3);
DEFINE_FILL_KERNEL(fill_u16_vec2, ushort, unsigned short, 2);
DEFINE_FILL_KERNEL(fill_u32_vec4, uint, unsigned int, 4);
DEFINE_FILL_KERNEL(fill_u32_vec3, uint, unsigned int, 3);
DEFINE_FILL_KERNEL(fill_u32_vec2, uint, unsigned int, 2);
DEFINE_FILL_KERNEL(fill_u64_vec4, ulonglong, unsigned long long, 4);
DEFINE_FILL_KERNEL(fill_u64_vec3, ulonglong, unsigned long long, 3);
DEFINE_FILL_KERNEL(fill_u64_vec2, ulonglong, unsigned long long, 2);
