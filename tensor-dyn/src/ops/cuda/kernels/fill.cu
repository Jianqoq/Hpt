
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define MAKE_VEC4(vec_type, vec_size, value) make_##vec_type##vec_size(value, value, value, value)
#define MAKE_VEC3(vec_type, vec_size, value) make_##vec_type##vec_size(value, value, value)
#define MAKE_VEC2(vec_type, vec_size, value) make_##vec_type##vec_size(value, value)

#define DEFINE_KERNEL(func_name, vec_type, type, vec_size)                            \
    extern "C" __global__ void func_name(type *out, type value, size_t N)             \
    {                                                                                 \
        vec_type##vec_size *out_vec = (vec_type##vec_size *)out;                      \
        vec_type##vec_size value_vec = MAKE_VEC##vec_size(vec_type, vec_size, value); \
                                                                                      \
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;                           \
        size_t stride = blockDim.x * gridDim.x;                                       \
        size_t N_vec = N / vec_size;                                                  \
                                                                                      \
        for (size_t i = idx; i < N_vec; i += stride)                                  \
        {                                                                             \
            out_vec[i] = value_vec;                                                   \
        }                                                                             \
                                                                                      \
        if (idx == 0)                                                                 \
        {                                                                             \
            for (size_t i = N_vec * vec_size; i < N; i++)                             \
            {                                                                         \
                out[i] = value;                                                       \
            }                                                                         \
        }                                                                             \
    }

extern "C" __global__ void fill_f16(half *out, half value, size_t N)
{
    __half2 *out_vec = (__half2 *)out;
    __half2 value_vec = make_half2(value, value);
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t N_vec = N / 2;
    for (size_t i = idx; i < N_vec; i += stride)
    {
        out_vec[i] = value_vec;
    }
    if (idx == 0)
    {
        for (size_t i = N_vec * 2; i < N; i++)
        {
            out[i] = value;
        }
    }
};
DEFINE_KERNEL(fill_f32, float, float, 4);
DEFINE_KERNEL(fill_f64, double, double, 4);

DEFINE_KERNEL(fill_i8, char, char, 4);
DEFINE_KERNEL(fill_i16, short, short, 4);
DEFINE_KERNEL(fill_i32, int, int, 4);
DEFINE_KERNEL(fill_i64, longlong, long long, 4);
DEFINE_KERNEL(fill_u8, uchar, unsigned char, 4);
DEFINE_KERNEL(fill_u16, ushort, unsigned short, 4);
DEFINE_KERNEL(fill_u32, uint, unsigned int, 4);
DEFINE_KERNEL(fill_u64, ulonglong, unsigned long long, 4);
