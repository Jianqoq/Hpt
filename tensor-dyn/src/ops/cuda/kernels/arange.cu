
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define MAKE_VEC4(vec_type, vec_size, value) make_##vec_type##vec_size(value, value + step, value + step * 2, value + step * 3)
#define MAKE_VEC3(vec_type, vec_size, value) make_##vec_type##vec_size(value, value + step, value + step * 2)
#define MAKE_VEC2(vec_type, vec_size, value) make_##vec_type##vec_size(value, value + step)

#define DEFINE_KERNEL(func_name, vec_type, type, vec_size)                      \
    extern "C" __global__ void func_name(type *out, type start, type step, size_t N) \
    {                                                                                \
        vec_type##vec_size *out_vec = (vec_type##vec_size *)out;                     \
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;                          \
        size_t stride = blockDim.x * gridDim.x;                                      \
        size_t N_vec = N / vec_size;                                                 \
                                                                                     \
        for (size_t i = idx; i < N_vec; i += stride)                                 \
        {                                                                            \
            type base = start + (i * vec_size) * step;                               \
            vec_type##vec_size value_vec = MAKE_VEC##vec_size(                       \
                vec_type, vec_size, base);                                           \
            out_vec[i] = value_vec;                                                  \
        }                                                                            \
                                                                                     \
        if (idx == 0)                                                                \
        {                                                                            \
            for (size_t i = N_vec * vec_size; i < N; i++)                            \
            {                                                                        \
                out[i] = start + i * step;                                           \
            }                                                                        \
        }                                                                            \
    }

extern "C" __global__ void arange_f16_vec2(half *out, half value, size_t N)
{
    __half2 *out_vec = (__half2 *)out;
    __half2 value_vec = make_half2(value, value);
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t N_vec = N / 2;
    for (size_t i = idx; i < N_vec; i += stride)
    {
        out_vec[i] = value_vec;
    }
    if (idx == 0)
    {
        for (size_t i = N_vec * 2; i < N; i++)
        {
            out[i] = value;
        }
    }
};
DEFINE_KERNEL(arange_f32_vec4, float, float, 4);
DEFINE_KERNEL(arange_f64_vec4, double, double, 4);

DEFINE_KERNEL(arange_i8_vec4, char, char, 4);
DEFINE_KERNEL(arange_i16_vec4, short, short, 4);
DEFINE_KERNEL(arange_i32_vec4, int, int, 4);
DEFINE_KERNEL(arange_i64_vec4, longlong, long long, 4);

DEFINE_KERNEL(arange_u8_vec4, uchar, unsigned char, 4);
DEFINE_KERNEL(arange_u16_vec4, ushort, unsigned short, 4);
DEFINE_KERNEL(arange_u32_vec4, uint, unsigned int, 4);
DEFINE_KERNEL(arange_u64_vec4, ulonglong, unsigned long long, 4);
